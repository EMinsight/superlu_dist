#include "hip/hip_runtime.h"
#include "superlu_ddefs.h"
#include "lupanels_GPU.cuh"
#include "lupanels.hpp"
// this should be a device code

// int_t lpanel_t::find(int_t k)
// {
//     for (int_t i = 0; i < nblocks(); i++)
//     {
//         if (k == gid(i))
//             return i;
//     }
//     //TODO: it shouldn't come here
//     return -1;
// }

//TODO: fix bug with syncthreads
__device__
int_t lpanelGPU_t::find(int_t k)
{
    int threadId = threadIdx.x;
    __shared__ int idx; 
    __shared__ int found;
    if(!threadId)
    {
        idx =-1;
        found=0;
    }

    int nThreads = blockDim.x; 
    int blocksPerThreads = CEILING( nblocks(),    nThreads);
    
    for(int blk =blocksPerThreads*threadIdx.x; 
        blk< blocksPerThreads*(threadIdx.x +1);
        blk++)
    {
        // if(found) break; 

        if(blk< nblocks())
        {
            if(k == gid(blk))
            {
                idx = blk;
                found =1;
            }
        }
        
    }
    __syncthreads();
    return idx;
}


__device__
int_t upanelGPU_t::find(int_t k)
{
    int threadId = threadIdx.x;
    __shared__ int idx; 
    __shared__ int found;
    if(!threadId)
    {
        idx =-1;
        found=0;
    }
        
    
    int nThreads = blockDim.x; 
    int blocksPerThreads = CEILING( nblocks(),    nThreads);
    
    for(int blk =blocksPerThreads*threadIdx.x; 
        blk< blocksPerThreads*(threadIdx.x +1);
        blk++)
    {
        // if(found) break; 

        if(blk< nblocks())
        {
            if(k == gid(blk))
            {
                idx = blk;
                found =1;
            }
        }
        
        
    }
    __syncthreads();
    return idx;
}


__device__
int computeIndirectMapGPU(int* rcS2D,  int_t srcLen, int_t *srcVec,
                                         int_t dstLen, int_t *dstVec,
                                         int *dstIdx)
{
    int threadId = threadIdx.x;
    if (dstVec == NULL) /*uncompressed dimension*/
    {
        if(threadId < srcLen)
            rcS2D[threadId] = srcVec[threadId];
        __syncthreads();
        return 0; 
    }
    
    
    if(threadId < dstLen)
        dstIdx[dstVec[threadId]] = threadId;
    __syncthreads();
    
    if(threadId < srcLen)
        rcS2D[threadId] = dstIdx[srcVec[threadId]];
    __syncthreads();
    
    return 0;
}

__global__
void scatterGPU(
    int iSt,  int jSt, 
    double* gemmBuff, int LDgemmBuff,
    lpanelGPU_t lpanel, upanelGPU_t upanel, 
    LUstructGPU_t* dA)
{

    // calculate gi,gj
    int ii = iSt + blockIdx.x; 
    int jj = jSt + blockIdx.y; 
    int threadId = threadIdx.x;

    int gi = lpanel.gid(ii);
    int gj = upanel.gid(jj);
    if(!threadId)
    printf("Scattering to (%d, %d) \n",gi, gj);
    double *Dst;
    int_t lddst;
    int_t dstRowLen, dstColLen;
    int_t *dstRowList;
    int_t *dstColList;
    
    if (gj > gi) // its in upanel
    {
        int li = dA->g2lRow(gi);
        int lj = dA->uPanelVec[li].find(gj);
        Dst = dA->uPanelVec[li].blkPtr(lj);
        // return; 
        lddst = dA->supersize(gi);
        dstRowLen = dA->supersize(gi);
        dstRowList = NULL;
        dstColLen = dA->uPanelVec[li].nbcol(lj);
        dstColList = dA->uPanelVec[li].colList(lj);
        // std::cout<<li<<" "<<lj<<" Dst[0] is"<<Dst[0] << "\n";
        if(!threadId)
        printf("Ui{j}k (%d, %d) \n",li, lj);
    }
    else
    {
        int lj = dA->g2lCol(gj);
        int li = dA->lPanelVec[lj].find(gi);
        Dst = dA->lPanelVec[lj].blkPtr(li);
        // return; 
        lddst = dA->lPanelVec[lj].LDA();
        dstRowLen = dA->lPanelVec[lj].nbrow(li);
        dstRowList = dA->lPanelVec[lj].rowList(li);
        dstColLen = dA->supersize(gj);
        dstColList = NULL;
        if(!threadId)
        printf("L{i}jk (%d, %d) \n",li, lj);
    }

    
    // compute source row to dest row mapping
    int maxSuperSize = dA->maxSuperSize; 
    extern __shared__ int baseSharedPtr[]; 
    int* rowS2D = baseSharedPtr;
    int* colS2D = &rowS2D[maxSuperSize];
    int* dstIdx = &colS2D[maxSuperSize];

    int nrows = lpanel.nbrow(ii);
    int ncols = upanel.nbcol(jj);
    // lpanel.rowList(ii), upanel.colList(jj)
    
    computeIndirectMapGPU(rowS2D,  nrows, lpanel.rowList(ii),
        dstRowLen, dstRowList, dstIdx);
    
    if(!threadId && !ii && !jj)
    {
        printf(" RowS2d nrows=%d ",nrows);
        for(int i=0; i< SUPERLU_MIN(5, nrows); i++)
            printf(" %d ",rowS2D[i]);
    }
        
    
// compute source col to dest col mapping
    
    computeIndirectMapGPU(colS2D, ncols, upanel.colList(jj),
        dstColLen, dstColList, dstIdx);
    
    if(!threadId && !ii && !jj)
    {
        printf(" ColS2d ncols=%d ",ncols);
        for(int i=0; i< SUPERLU_MIN(5, ncols); i++)
            printf(" %d ",colS2D[i]);
    }

    int nThreads = blockDim.x; 
    int colsPerThreadBlock = nThreads/ nrows;
    
    
    

    int rowOff = lpanel.stRow(ii) - lpanel.stRow(iSt);
    int colOff = upanel.stCol(jj) - upanel.stCol(jSt);
    double* Src = &gemmBuff[ rowOff+ colOff* LDgemmBuff];
    int ldsrc = LDgemmBuff; 
    // TODO: this seems inefficient 
    if (threadId < nrows * colsPerThreadBlock)
	{
		/* 1D threads are logically arranged in 2D shape. */
		int i = threadId % nrows;
		int j = threadId / nrows;

        #pragma unroll 4
        while(j<ncols)
        {   
            if(gi==521 && gj==521 && i+j<5)
                printf(" (%d %d, %lf %lf)\n", i, j, Dst[rowS2D[i] + lddst * colS2D[j]], Src[i + ldsrc * j]);

            Dst[rowS2D[i] + lddst * colS2D[j]] -= Src[i + ldsrc * j];
            j += colsPerThreadBlock;
        }
		
	}

    __syncthreads();
} 


int_t LUstruct_v100::dSchurComplementUpdateGPU(
    int streamId, 
    int_t k, lpanel_t &lpanel, upanel_t &upanel)
{
    // TODO: redefine isEmpty so this works out 
    if (lpanel.isEmpty() || upanel.isEmpty())
        return 0;

    int_t st_lb = 0;
    if (myrow == krow(k))
        st_lb = 1;

    int_t nlb = lpanel.nblocks();
    int_t nub = upanel.nblocks();

    int iSt =st_lb;
    int iEnd =iSt; 
    
    

    int nrows = lpanel.stRow(nlb) - lpanel.stRow(st_lb);
    int ncols = upanel.nzcols(); 

    int maxGemmRows = nrows;  
    int maxGemmCols = ncols;
    // entire gemm doesn't fit in gemm buffer
    if(nrows* ncols > A_gpu.gemmBufferSize)
    {
        int maxGemmOpSize = (int) sqrt(A_gpu.gemmBufferSize);
        int numberofRowChunks = (nrows +maxGemmOpSize-1)/maxGemmOpSize;
        maxGemmRows =   nrows /numberofRowChunks;
        maxGemmCols = A_gpu.gemmBufferSize/ maxGemmRows; 
    }
    
    while(iEnd< nlb)
    {
        iSt = iEnd;
        iEnd = lpanel.getEndBlock(iSt, maxGemmRows);
        
        assert(iEnd>iSt);
        int jSt =0;
        int jEnd =0; 
        while(jEnd< nub)
        {
            jSt = jEnd; 
            jEnd = upanel.getEndBlock(jSt, maxGemmCols);
            assert(jEnd>jSt);
            hipblasHandle_t handle = A_gpu.cuHandles[streamId];
            hipStream_t cuStream = A_gpu.cuStreams[streamId];
            hipblasSetStream(handle, cuStream);
            int gemm_m = lpanel.stRow(iEnd) - lpanel.stRow(iSt);
            int gemm_n = upanel.stCol(jEnd) - upanel.stCol(jSt);
            int gemm_k = supersize(k);
            double alpha = 1.0;
            double beta = 0.0; 
            
            printf("m=%d, n=%d, k=%d\n", gemm_m,gemm_n,gemm_k);
            hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                        gemm_m, gemm_n, gemm_k, &alpha,
                        lpanel.blkPtrGPU(iSt), lpanel.LDA(),
                        upanel.blkPtrGPU(jSt), upanel.LDA(), &beta,
                        A_gpu.gpuGemmBuffs[streamId], gemm_m);


            // setting up scatter 
            dim3 dimBlock(ldt); // 1d thread
            dim3 dimGrid(iEnd - iSt, jEnd - jSt);
            size_t sharedMemorySize=3* A_gpu.maxSuperSize * sizeof(int_t); 

            scatterGPU<<<dimGrid, dimBlock, sharedMemorySize, cuStream>>>(
                iSt, jSt, 
                A_gpu.gpuGemmBuffs[streamId], gemm_m,
                lpanel.gpuPanel, upanel.gpuPanel, dA_gpu);   
            
            
		}
    }
    return 0;
}




