#include "hip/hip_runtime.h"
/*! \file
Copyright (c) 2003, The Regents of the University of California, through
Lawrence Berkeley National Laboratory (subject to receipt of any required
approvals from U.S. Dept. of Energy)

All rights reserved.

The source code is distributed under BSD license, see the file License.txt
at the top-level directory.
*/


/*! @file
 * \brief Solves a system of distributed linear equations A*X = B with a
 * general N-by-N matrix A using the LU factors computed previously.
 *
 * <pre>
 * -- Distributed SuperLU routine (version 6.1) --
 * Lawrence Berkeley National Lab, Univ. of California Berkeley.
 * October 15, 2008
 * September 18, 2018  version 6.0
 * February 8, 2019  version 6.1.1
 * </pre>
 */

 #include <math.h> 
 #include "superlu_ddefs.h"
 #ifndef CACHELINE
 #define CACHELINE 64  /* bytes, Xeon Phi KNL, Cori haswell, Edision */
 #endif
 #include <stdio.h>
 #include "mpi.h"
#include <nvshmem.h>
#include <nvshmemx.h>
 #include <stdlib.h>
 #include <sched.h>
 #include <nvml.h>
 #include <omp.h>


 #undef CUDA_CHECK
 #define CUDA_CHECK(stmt)                                                          \
     do {                                                                          \
         hipError_t result = (stmt);                                              \
         if (hipSuccess != result) {                                              \
             fprintf(stderr, "[%s:%d] cuda failed with %s \n", __FILE__, __LINE__, \
                     hipGetErrorString(result));                                  \
             exit(-1);                                                             \
         }                                                                         \
         assert(hipSuccess == result);                                            \
     } while (0)

 #undef MPI_CHECK
 #define MPI_CHECK(stmt)                                 \
 do {                                                    \
     int result = (stmt);                                \
     if (MPI_SUCCESS != result) {                        \
         fprintf(stderr, "[%s:%d] MPI failed with error %d \n",\
          __FILE__, __LINE__, result);                   \
         exit(-1);                                       \
     }                                                   \
 } while (0)

 #define NVSHMEM_CHECK(stmt)                               \
 do {                                                    \
     int result = (stmt);                                \
     if (hipSuccess != result) {                      \
         fprintf(stderr, "[%s:%d] nvshmem failed with error %d \n",\
          __FILE__, __LINE__, result);                   \
         exit(-1);                                       \
     }                                                   \
 } while (0)



#ifdef __cplusplus
	extern "C" {
#endif
 
	 
 // #define USESHARE1RHS 1	
	 
	 
 /***************************************************************************//**
	 Does sum reduction of n-element array x, leaving total in x[0].
	 Contents of x are destroyed in the process.
	 With k threads, can reduce array up to 2*k in size.
	 Assumes number of threads <= 1024 (which is max number of threads up to CUDA capability 3.0)
	 Having n as template parameter allows compiler to evaluate some conditions at compile time.
	 Calls __syncthreads before & after reduction.
	 @ingroup magma_kernel
 *******************************************************************************/
 __device__ void
 magma_sum_reduce( int n, int i, double* x )
 {
	 __syncthreads();
	 if ( n > 1024 ) { if ( i < 1024 && i + 1024 < n ) { x[i] += x[i+1024]; }  __syncthreads(); }
	 if ( n >  512 ) { if ( i <  512 && i +  512 < n ) { x[i] += x[i+ 512]; }  __syncthreads(); }
	 if ( n >  256 ) { if ( i <  256 && i +  256 < n ) { x[i] += x[i+ 256]; }  __syncthreads(); }
	 if ( n >  128 ) { if ( i <  128 && i +  128 < n ) { x[i] += x[i+ 128]; }  __syncthreads(); }
	 if ( n >   64 ) { if ( i <   64 && i +   64 < n ) { x[i] += x[i+  64]; }  __syncthreads(); }
	 if ( n >   32 ) { if ( i <   32 && i +   32 < n ) { x[i] += x[i+  32]; }  __syncthreads(); }
	 // probably don't need __syncthreads for < 16 threads
	 // because of implicit warp level synchronization.
	 if ( n >   16 ) { if ( i <   16 && i +   16 < n ) { x[i] += x[i+  16]; }  __syncthreads(); }
	 if ( n >    8 ) { if ( i <    8 && i +    8 < n ) { x[i] += x[i+   8]; }  __syncthreads(); }
	 if ( n >    4 ) { if ( i <    4 && i +    4 < n ) { x[i] += x[i+   4]; }  __syncthreads(); }
	 if ( n >    2 ) { if ( i <    2 && i +    2 < n ) { x[i] += x[i+   2]; }  __syncthreads(); }
	 if ( n >    1 ) { if ( i <    1 && i +    1 < n ) { x[i] += x[i+   1]; }  __syncthreads(); }
 }
 // end sum_reduce
 

	 
 /******************************************************************************/
 static __device__ void
 gemv_device_dlsum_fmod(
	 int_t m, int_t n, double alpha,
	 const double * __restrict__ A, int_t lda,
	 const double * __restrict__ x, int_t incx, double beta,
	 double       * __restrict__ y, int_t incy)
 {
	 if (m <= 0 || n <= 0) return;
 
	 int_t num_threads = DIM_X * DIM_Y;
	 int_t thread_id = threadIdx_x + threadIdx_y * blockDim_x;
 
	 // threads are all configurated locally
	 int_t tx = thread_id % DIM_X;
	 int_t ty = thread_id / DIM_X;
 
	 int_t ind = tx;
 
	 __shared__ double sdata[DIM_X * DIM_Y];
 
 
	 int_t st = 0;
 
	 int_t ed = min(st+m, CEILING(m,DIM_X)*DIM_X);
	 
	 int_t iters = CEILING(ed-st,DIM_X) ;
 
	 double zero = 0.0;
	 
	 for (int_t i=0; i < iters; i++)
	 {   
		 if (ind < m ) A += ind;
 
		 double res = zero;
		 
		 if (ind < m )
		 {
			 for (int_t col=ty; col < n; col += DIM_Y)
			 {       
				 res += A[col*lda] * x[col*incx];
			 }
		 }
 
		 if (DIM_X >= num_threads) // indicated 1D threads configuration. Shared memory is not needed, reduction is done naturally
		 {
			 if (ty == 0 && ind < m)
			 {
				 y[ind*incy] = alpha*res + beta*y[ind*incy];
			 }
		 }
		 else 
		 {
			 sdata[ty + tx * DIM_Y] = res;
 
			 __syncthreads(); 
 
			 if ( DIM_Y > 16)
			 { 
				 magma_sum_reduce(DIM_Y, ty, sdata + tx * DIM_Y);
			 }
			 else
			 {
				 if (ty == 0 && ind < m)
				 {
					 for (int_t i=1; i < DIM_Y; i++)
					 {
						 sdata[tx * DIM_Y] += sdata[i + tx * DIM_Y]; 
					 }
				 }
			 }
 
			 if (ty == 0 && ind < m)
			 {
				 y[ind*incy] = alpha*sdata[tx * DIM_Y] + beta*y[ind*incy];
			 }
 
			 __syncthreads();
		 }
 
		 if ( ind < m) A -= ind;
 
		 ind += DIM_X;
	 }
 }
 
	 
	 
	 
 
 /******************************************************************************/
 static __device__ 
 void gemm_device_dlsum_fmod(
	 int_t M, int_t N, int_t K,
	 int_t blx, int_t bly,
	 const double* __restrict__ A, int_t LDA,
	 const double* __restrict__ B, int_t LDB,
	 double rC[THR_N][THR_M],
	 double alpha, double beta)
 {
 // #if (__CUDA_ARCH__ >= 200)
	 int_t idx = threadIdx_x;  // thread's m dimension
	 int_t idy = threadIdx_y;  // thread's n dimension
 
	 int_t idt = DIM_X * idy + idx;    // thread's global number
 
	 int_t idxA = idt % DIM_XA;    // idx within A
	 int_t idyA = idt / DIM_XA;    // idy within A
 
	 int_t idxB = idt % DIM_XB;    // idx within B
	 int_t idyB = idt / DIM_XB;    // idy within B
 
	 // int_t blx = blockIdx_x;   // block's m dimension
	 // int_t bly = blockIdx_y;   // block's n dimension
 
	 __shared__ double sA[BLK_K][BLK_M+1];      // +1 only required if A is transposed
	 __shared__ double sB[BLK_N][BLK_K+1];      // +1 always required	
	 
	 // Registers for the innermost loop
	 double rA[THR_M];
	 double rB[THR_N];
 
	 double ra[BLK_K/DIM_YA+1][BLK_M/DIM_XA];
	 double rb[BLK_N/DIM_YB][BLK_K/DIM_XB+1];
	 
	 const double *offs_dA = A + blx*BLK_M     + idyA*LDA + idxA;
	 const double *offs_dB = B + bly*BLK_N*LDB + idyB*LDB + idxB;
	 int_t boundA = (LDA*(K-1) + M) - ( blx*BLK_M  + idyA*LDA + idxA ) -1;
	 int_t boundB = (LDB*(N-1) + K) - ( bly*BLK_N*LDB + idyB*LDB + idxB ) -1;
 
	 int_t m, n, k, kk;
	 double zero = 0.0;
 
	 // Zero C
	 #pragma unroll
	 for (n = 0; n < THR_N; n++)
		 #pragma unroll
		 for (m = 0; m < THR_M; m++)
			 rC[n][m] = zero;
 
	 #pragma unroll
	 for (n = 0; n < BLK_K; n += DIM_YA)
		 #pragma unroll
		 for (m = 0; m < BLK_M; m += DIM_XA)
			 sA[n+idyA][m+idxA] = fetch(A, m, n, boundA);
	 
	 #pragma unroll
	 for (n = 0; n < BLK_N; n += DIM_YB)
		 #pragma unroll
		 for (m = 0; m < BLK_K; m += DIM_XB)
			 sB[n+idyB][m+idxB] = fetch(B, m, n, boundB);
	 
	 __syncthreads();
 
	 for (kk = 0; kk < K-BLK_K; kk += BLK_K)
	 {
		 offs_dA += BLK_K*LDA;
		 boundA  -= BLK_K*LDA;
 
		 offs_dB += BLK_K;
		 boundB  -= BLK_K;
 
		 #pragma unroll
		 for (n = 0; n < BLK_K/DIM_YA; n++)
			 #pragma unroll
			 for (m = 0; m < BLK_M/DIM_XA; m++)
				 ra[n][m] = fetch(A, m*DIM_XA, n*DIM_YA, boundA);
 
		 #pragma unroll
		 for (n = 0; n < BLK_N/DIM_YB; n++)
			 #pragma unroll
			 for (m = 0; m < BLK_K/DIM_XB; m++)
				 rb[n][m] = fetch(B, m*DIM_XB, n*DIM_YB, boundB);
		 
		 // Multiply
		 #pragma unroll
		 for (k = 0; k < BLK_K; k++)
		 {
			 // Load A shmem->regs
			 #pragma unroll
			 for (m = 0; m < THR_M; m++)
				 rA[m] = sA[k][m*DIM_X+idx];
 
			 // Load B shmem->regs
			 #pragma unroll
			 for (n = 0; n < THR_N; n++)
				 rB[n] = sB[n*DIM_Y+idy][k];
 
			 // Compute
			 #pragma unroll
			 for (n = 0; n < THR_N; n++) {
				 #pragma unroll
				 for (m = 0; m < THR_M; m++) {
					 fma(rA[m], rB[n], rC[n][m]);
				 }
			 }
		 }
 
		 __syncthreads();
 
		 #pragma unroll
		 for (n = 0; n < BLK_K/DIM_YA; n++)
			 #pragma unroll
			 for (m = 0; m < BLK_M/DIM_XA; m++)
				 sA[n*DIM_YA+idyA][m*DIM_XA+idxA] = ra[n][m];
		 
		 #pragma unroll
		 for (n = 0; n < BLK_N/DIM_YB; n++)
			 #pragma unroll
			 for (m = 0; m < BLK_K/DIM_XB; m++)
				 sB[n*DIM_YB+idyB][m*DIM_XB+idxB] = rb[n][m];
		 
		 __syncthreads();
	 }
 
	 // Multiply last full (BLK_K) or partial block of
	 // columns of op(A) and rows of op(B).
	 // It's okay that m,n exceed matrix bounds as all work is in registers
	 // or shared memory, and out-of-bounds rC[n][m] will not be saved later.
	 kk = K - kk;
	 #pragma unroll
	 for (k = 0; k < kk; k++)
	 {
		 // Load A shmem->regs
		 #pragma unroll
		 for (m = 0; m < THR_M; m++)
			 rA[m] = sA[k][m*DIM_X+idx];
 
		 // Load B shmem->regs
		 #pragma unroll
		 for (n = 0; n < THR_N; n++)
			 rB[n] = sB[n*DIM_Y+idy][k];
 
		 // Compute
		 #pragma unroll
		 for (n = 0; n < THR_N; n++) {
			 #pragma unroll
			 for (m = 0; m < THR_M; m++) {
				 fma(rA[m], rB[n], rC[n][m]);
			 }
		 }
	 }
 
	 // Store C regs->dev
	 // if( beta == make_FloatingPoint_t(0.0,0.0) ) {
		 // #pragma unroll
		 // for (n = 0; n < THR_N; n++) {
			 // int_t coord_dCn = bly*BLK_N + n*DIM_Y + idy;
			 // #pragma unroll
			 // for (m = 0; m < THR_M; m++) {
				 // int_t coord_dCm = blx*BLK_M + m*DIM_X + idx;
				 // if (coord_dCm < M && coord_dCn < N) {
					 // int_t offsC = coord_dCn*LDC + coord_dCm;
 
					 // double &regC = rC[n][m];
					 // double &memC = C[offsC];
 
					 // // memC = mul(alpha, regC);
				 // }
			 // }
		 // }
	 // } else {
		 // #pragma unroll
		 // for (n = 0; n < THR_N; n++) {
			 // int_t coord_dCn = bly*BLK_N + n*DIM_Y + idy;
			 // #pragma unroll
			 // for (m = 0; m < THR_M; m++) {
				 // int_t coord_dCm = blx*BLK_M + m*DIM_X + idx;
				 // if (coord_dCm < M && coord_dCn < N) {
					 // int_t offsC = coord_dCn*LDC + coord_dCm;
 
					 // double &regC = rC[n][m];
					 // double &memC = C[offsC];
 
					 // // memC = add(mul(alpha, regC), mul(beta, memC));
				 // }
			 // }
		 // }
	 // }
 // #endif /* (__CUDA_ARCH__ >= 200) */
 }
#define cudaCheckError() { \
    hipError_t e=hipGetLastError();                           \
    if(e!=hipSuccess) {                       \
        printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));                           \
        exit(EXIT_FAILURE);                   \
    }                       \
}
void checkP2Paccess(int numGPUs)
{
    for (int i = 0; i < numGPUs; i++) {
        hipSetDevice(i);
        cudaCheckError();

        for (int j = 0; j < numGPUs; j++) {
            int access;
            if (i != j) {
                hipDeviceCanAccessPeer(&access, i, j);
                cudaCheckError();
                printf("Device=%d %s Access Peer Device=%d\n", i, access ? "CAN" : "CANNOT", j);
            }
        }
    }
    printf("\n***NOTE: In case a device doesn't have P2P access to other one, it falls back to normal memcopy procedure.\nSo you can see lesser Bandwidth (GB/s) and unstable Latency (us) in those cases.\n\n");
}

void check_connectivity(){
//Check peer-to-peer connectivity
    int numGPUs;
    hipGetDeviceCount(&numGPUs);
    cudaCheckError();

    for (int i = 0; i < numGPUs; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        cudaCheckError();
        printf("Device: %d, %s, pciBusID: %x, pciDeviceID: %x, pciDomainID:%x\n", i, prop.name, prop.pciBusID, prop.pciDeviceID, prop.pciDomainID);
    }
    printf("P2P Connectivity Matrix\n");
    printf("     D\\D");
    for (int j = 0; j < numGPUs; j++) {
        printf("%6d", j);
    }
    printf("\n");

    for (int i = 0; i < numGPUs; i++) {
        printf("%6d\t", i);
        for (int j = 0; j < numGPUs; j++) {
            if (i != j) {
                int access;
                hipDeviceCanAccessPeer(&access, i, j);
                cudaCheckError();
                printf("%6d", (access) ? 1 : 0);
            }else {
                printf("%6d", 1);
            }
        }
        printf("\n");
    }

 }

void nv_init_wrapper(int* c, char *v[], int* omp_mpi_level)
{
    int *target;
    int rank, nranks, ndevices;
    MPI_Comm mpi_comm;
    nvshmemx_init_attr_t attr;
    int mype, npes, mype_node;
    //MPI_CHECK(MPI_Init(&c, &v));
    MPI_CHECK(MPI_Init_thread( c, &v, MPI_THREAD_MULTIPLE, omp_mpi_level));
    MPI_CHECK(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    MPI_CHECK(MPI_Comm_size(MPI_COMM_WORLD, &nranks));

    mpi_comm = MPI_COMM_WORLD;
    attr.mpi_comm = &mpi_comm;

    NVSHMEM_CHECK(nvshmemx_init_attr (NVSHMEMX_INIT_WITH_MPI_COMM, &attr));
    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    CUDA_CHECK(hipSetDevice(mype_node));

    //char name[MPI_MAX_PROCESSOR_NAME];
    //int resultlength;
    //MPI_CHECK(MPI_Get_processor_name(name, &resultlength));
    //int get_cur_dev;
    //CUDA_CHECK(hipGetDevice(&get_cur_dev));
    //CUDA_CHECK(hipGetDeviceCount(&ndevices));
    //printf("** MPI %d/%d, NVSHMEM %d/%d , ndevices=%d,cur=%d, node=%s **\n",rank,nranks,mype,npes,ndevices,get_cur_dev,name);
    //fflush(stdout);

}


__device__ void C_BcTree_forwardMessageSimple_Device(C_Tree* tree,  int* flag_bc_q,  int* my_flag_bc, int mype, int tid,double* ready_x){
    //int BCsendoffset;
    for( int idxRecv = 0; idxRecv < tree->destCnt_; ++idxRecv ) {
        int iProc = tree->myDests_[idxRecv];
        //BCsendoffset = my_flag_bc[2];
        double sum=0;
        if (tid==0) {
            for(int i=0;i<my_flag_bc[3];i++){
                //printf("(%d), data, %d,%lf\n",mype,i,ready_x[i]);
                sum+=ready_x[i];
            }
            printf("Start (%d), forwardDevice, send to %d, signal offset=%d, msgsz=%d,sum=%lf\n",mype,iProc,my_flag_bc[1],my_flag_bc[3],sum);
        }
        //__syncthreads();
        //if(tid==0) printf("Start (%d), forwardDevice, send to %d, signal offset=%d, data offset=%d, msgsz=%d\n",mype,iProc,my_flag_bc[1],BCsendoffset,my_flag_bc[3]);
        //__syncthreads();
        //nvshmemx_double_put_block(&ready_x[BCsendoffset],ready_x,my_flag_bc[3],iProc);
        nvshmemx_double_put_nbi_block(ready_x, &ready_x[0], my_flag_bc[3], iProc);
        //nvshmem_double_put_nbi(ready_x, &ready_x[0], my_flag_bc[3], iProc);
        //nvshmem_double_put(&ready_x[BCsendoffset],ready_x,my_flag_bc[3],iProc);
        //nvshmem_quiet();
        nvshmem_fence();
        //__syncthreads();
        if (tid == 0) {
            int sig = 1;
            nvshmemx_int_signal((int*)(flag_bc_q + my_flag_bc[1]), sig, iProc);
            //nvshmem_quiet();
            printf("Done (%d), forwardDevice, send to %d, signal offset=%d, data offset=%d, msgsz=%d\n", mype, iProc,
                    my_flag_bc[1], my_flag_bc[2], my_flag_bc[3]);

        }
    }
}

__device__ void C_RdTree_forwardMessageSimple_Device(C_Tree* Tree, int* flag_rd_q, int* my_flag_rd, int mype, int bid, int tid, double* ready_lsum, int maxrecvsz){
    int data_ofset,sig_ofset;
    if(Tree->myIdx %2 ==0){
        sig_ofset = my_flag_rd[0]*2;
        data_ofset = my_flag_rd[0]*maxrecvsz*2;
    }else{
        sig_ofset = my_flag_rd[0]*2+1;
        data_ofset = my_flag_rd[0]*maxrecvsz*2+maxrecvsz;
    }
	if(Tree->myRank_!=Tree->myRoot_){
		  ////forward to my root if I have received everything
		int iProc = Tree->myRoot_;
		  //// YL: Use NVSHMEM to send to my parent
        printf("forwardMessage-(%d,%d,%d), rd--1, send to %d,data_ofset=%d,maxrecvsz=%d,k=%d,sigoffset=%d\n",mype,bid, tid,iProc,data_ofset,maxrecvsz,my_flag_rd[0],sig_ofset);
        //__device__ void nvshmem_TYPENAME_put(TYPE *dest, const TYPE *source, size_t nelems, int pe)
        double sum=0;
        for(int i=my_flag_rd[0]*maxrecvsz*2;i<my_flag_rd[0]*maxrecvsz*2+my_flag_rd[1];i++){
            //printf("(%d), data, %d,%lf\n",mype,i,ready_lsum[i]);
            sum+=ready_lsum[i];
        }
        printf("forwardMessage- (%d), forwardDevice, send to %d, lib=%d,sum=%lf\n",mype,iProc,my_flag_rd[0],sum);

        nvshmem_double_put(&ready_lsum[data_ofset],&ready_lsum[my_flag_rd[0]*maxrecvsz*2],my_flag_rd[1],iProc);
        nvshmem_fence();
        int sig=1;
        nvshmemx_int_signal((int*)flag_rd_q+sig_ofset, sig, iProc);
        printf("forwardMessage-(%d,%d,%d), rd--2, send to %d,data_ofset=%d,sig_ofset=%d\n",mype,bid, tid,iProc,data_ofset,sig_ofset);
	}
}


__global__ void schedule
(
int nrhs,
int_t nbcol_loc,
C_Tree  *LBtree_ptr,
C_Tree  *LRtree_ptr,
int_t maxrecvsz,
int mype,
int* flag_bc_q,
int* flag_rd_q,
double* ready_x,
double* ready_lsum,
int* my_flag_bc,
int* my_flag_rd,
int totalth,
int* d_nfrecv,
int* d_status,
int* d_launch_flag,
int* d_colnum,
int* d_mynum,
int* d_mymaskstart,
int* d_mymasklength,
int* d_nfrecvmod,
int* d_statusmod,
int* d_colnummod,
int* d_mynummod,
int* d_mymaskstartmod,
int* d_mymasklengthmod,
int* d_recv_cnt,
int* d_msgnum
)
{
    int bid = blockIdx.x;
    int global_id = blockIdx.x * blockDim.x * blockDim.y + threadIdx.x + threadIdx.y * blockDim.x;
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    d_launch_flag[0]=1;
    int WAIT_NUM_THREADS=d_nfrecv[1];
    //printf("(%d,%d,%d),global_id=%d,WAIT_NUM_THREADS=%d\n",mype,bid,tid,global_id,WAIT_NUM_THREADS);
    //if(bid==1) printf("(%d,%d,%d),orz\n",mype,bid,tid);
    if (bid==1) { // for RD
        //if(tid==0){
        //    //for(int i=0;i<15;i++) {
        //    //        printf("(%d) recv_cnt[%d]=%d,%d,"
        //    //               "flag[%d,%d]=%d,%d,"
        //    //               "status=%d,%d\n", mype, i, d_recv_cnt[i], d_nfrecvmod[0],
        //    //               i * 2, i * 2 + 1, flag_rd_q[i * 2], flag_rd_q[i * 2 + 1],
        //    //               d_statusmod[i * 2], d_statusmod[i * 2 + 1]);

        //    //}
        //    //printf("(%d,%d,%d),waitcol=%d,wait msg=%d\n",mype,bid,tid,d_nfrecvmod[1],d_nfrecvmod[0]);
        //    for (int i=0;i<d_nfrecvmod[1];i++){
        //        printf("(%d),waitcol=%d,col[%d]=%d\n",mype,d_nfrecvmod[1],i,d_colnummod[i]);
        //    }
        //}
        //__syncthreads();
        if (WAIT_NUM_THREADS >= d_nfrecvmod[1]) { // one thread wait for one col
            if (tid < d_nfrecvmod[1]) {
                //printf("(%d,%d,%d) d_colnummod=%d,recv_cnt=%d\n", mype, bid, tid, d_colnummod[tid], d_recv_cnt[d_colnummod[tid]]);
                for (int i = 0; i < d_recv_cnt[d_colnummod[tid]]; i++) {
                    printf("(%d,%d,%d) d_colnummod=%d,recv_cnt=%d,i=%d,wait_off=%d,%d,status=%d,%d\n", mype, bid, tid, d_colnummod[tid], d_recv_cnt[d_colnummod[tid]],i,d_colnummod[tid]*2, d_colnummod[tid]*2+1,d_statusmod[d_colnummod[tid]*2], d_statusmod[d_colnummod[tid]*2+1]);
                    int wm_val = nvshmem_int_wait_until_any(flag_rd_q + d_colnummod[tid]*2, 2,
                                                            d_statusmod + d_colnummod[tid]*2, NVSHMEM_CMP_EQ, 1);
                    d_statusmod[d_colnummod[tid] * 2 + wm_val] = 1;
                    //printf("(%d,%d,%d) d_colnummod=%d,done_off=%d,status=%d,flag_rd_q=%d\n", mype, bid, tid, d_colnummod[tid], d_colnummod[tid]*2+wm_val, d_statusmod[d_colnummod[tid]*2+wm_val],flag_rd_q[d_colnummod[tid]*2+wm_val] );
                }
            }
        } else {
            int delta = d_nfrecvmod[1] % WAIT_NUM_THREADS;
            //d_mynummod: #col I wait.
            if (tid < delta) {
                d_mynummod[tid] = d_nfrecvmod[1] / WAIT_NUM_THREADS + 1;
            } else {
                d_mynummod[tid] = d_nfrecvmod[1] / WAIT_NUM_THREADS;
            }
            __syncthreads();

            d_mymaskstartmod[tid] = 0;
            d_msgnum[tid]=0;

            //d_mymaskstartmod: start offset of d_colnummod
            for (int i = 0; i < tid; i++) {
                d_mymaskstartmod[tid] += d_mynummod[i];
                //printf("(%d,%d,%d),i=%d,d_mynummod=%d,d_mymaskstartmod=%d\n",
                //       mype,bid,tid,i,
                //       d_mynummod[i],d_mymaskstartmod[tid]);
            }
            d_mymasklengthmod[tid] = 2* (d_colnummod[d_mymaskstartmod[tid] + d_mynummod[tid] - 1]
                                     - d_colnummod[d_mymaskstartmod[tid]] + 1);
            __syncthreads();
            for (int i=d_mymaskstartmod[tid];i<d_mymaskstartmod[tid]+d_mynummod[tid];i++){
                d_msgnum[tid]+= d_recv_cnt[d_colnummod[i]];
                //printf("(%d,%d,%d),i=%d,d_recv_cnt=%d\n",mype,bid,tid,i,d_recv_cnt[d_colnummod[i]]);
            }
            //printf("(%d,%d,%d) waitcol=%d,msgnum=%d,masklength=%d,start=%d\n",mype,bid,tid,d_mynummod[tid],d_msgnum[tid],d_mymasklengthmod[tid],d_mymaskstartmod[tid]);
            for (int i = 0; i < d_msgnum[tid]; i++) {
                int wm_val = nvshmem_int_wait_until_any(flag_rd_q + d_colnummod[d_mymaskstartmod[tid]]*2,
                                                        d_mymasklengthmod[tid],
                                                        d_statusmod + d_colnummod[d_mymaskstartmod[tid]]*2,
                                                        NVSHMEM_CMP_EQ, 1);
                d_statusmod[d_colnummod[d_mymaskstartmod[tid]] + wm_val] = 1;
            }
        }
    }



    if (bid==0) { // for BC
        //if(tid==0) printf("(%d,%d) ****BC******\n",mype,bid);
        if (WAIT_NUM_THREADS >= d_nfrecv[0]) {
            if (tid >= d_nfrecv[0]) return;
            nvshmem_int_wait_until(flag_bc_q + d_colnum[tid], NVSHMEM_CMP_EQ, 1);
            d_status[d_colnum[tid]] = 1;
        } else {
            int delta = d_nfrecv[0] % WAIT_NUM_THREADS;
            if (tid < delta) {
                d_mynum[tid] = d_nfrecv[0] / WAIT_NUM_THREADS + 1;
            } else {
                d_mynum[tid] = d_nfrecv[0] / WAIT_NUM_THREADS;
            }
            __syncthreads();
            d_mymaskstart[tid] = 0;
            for (int i = 0; i < tid; i++) {
                d_mymaskstart[tid] += d_mynum[i];
            }
            d_mymasklength[tid] = d_colnum[d_mymaskstart[tid] + d_mynum[tid] - 1] - d_colnum[d_mymaskstart[tid]] + 1;
            for (int i = 0; i < d_mynum[tid]; i++) {
                int wm_val = nvshmem_int_wait_until_any(flag_bc_q + d_colnum[d_mymaskstart[tid]], d_mymasklength[tid],
                                                        d_status + d_colnum[d_mymaskstart[tid]], NVSHMEM_CMP_EQ, 1);
                d_status[d_colnum[d_mymaskstart[tid]] + wm_val] = 1;
            }
        }
    }


}

//  /************************************************************************/
//  /*! \brief
//   *
//   * <pre>
//   * Purpose
//   * =======
//   *   Perform local block modifications: lsum[i] -= L_i,k * X[k].
//   * </pre>
//   */
//  __global__ void dlsum_fmod_inv_gpu_1rhs
//  /************************************************************************/
//  (
//   double *lsum,    /* Sum of local modifications.                        */
//   double *x,       /* X array (local)                                    */
//   double *rtemp,   /* Result of full matrix-vector multiply.             */
//   int   nrhs,      /* Number of right-hand sides.                        */
//   int   maxsup,      /* Max supernode size.                        */
//   int_t   nsupers,      /* Number of total supernodes.                        */
//   int_t *fmod,     /* Modification count for L-solve.                    */
//   int_t *xsup,
//   gridinfo_t *grid,
//   LocalLU_t *Llu
//  )
//  {
// 	 double alpha = 1.0, beta = 0.0,malpha=-1.0;
// 	 double *lusup, *lusup1;
// 	 double *dest;
// 	 double *Linv;/* Inverse of diagonal block */
// 	 int    iam, iknsupc, myrow, mycol, krow, nbrow, nbrow1, nbrow_ref, nsupr, nsupr1, p, pi, idx_r,m;
// 	 int_t  k,i, l,ii,jj, ik, il, ikcol, irow, j, lb, lk, rel, lib,lready;
// 	 int_t  *lsub, *lsub1, nlb1, lptr1, luptr1,*lloc;
// 	 int_t  luptr_tmp,luptr_tmp1,lptr1_tmp,maxrecvsz, idx_i, idx_v,idx_n,  idx_l, fmod_tmp, lbstart,lbend,nn,Nchunk,nlb_loc,remainder;
// 	 int thread_id1;
// 	 flops_t ops_loc=0.0;
// 	 MPI_Status status;
// 	 int test_flag;
// 	 yes_no_t done;
// 	 C_Tree  *LBtree_ptr = Llu->LBtree_ptr;
// 	 C_Tree  *LRtree_ptr = Llu->LRtree_ptr;
// 	 int_t* idx_lsum,idx_lsum1;
// 	 const int Nbk=1;
	 
//  #ifdef USESHARE1RHS	
// 	 const int MaxSUP=128;  // warning: this is the maximum size of supernodes, currently hardcoded
// 	 // __shared__ double rtemp_loc[128]; 
// 	 volatile __shared__ int s_fmod[NWARP];
// 	 volatile __shared__ double s_lsum[NWARP*MaxSUP];
//  #endif	
// 	 double temp,temp1;
// 	 int_t ldalsum;
// 	 int_t nleaf_send_tmp;
// 	 int_t lptr;      /* Starting position in lsub[*].                      */
// 	 int_t luptr;     /* Starting position in lusup[*].                     */
// 	 int_t iword = sizeof(int_t);
// 	 int_t dword = sizeof (double);
// 	 int_t aln_d,aln_i;
// 	 aln_d = 1;//ceil(CACHELINE/(double)dword);
// 	 aln_i = 1;//ceil(CACHELINE/(double)iword);
// 	 int   knsupc;    /* Size of supernode k.                               */
// 	 int_t nlb;       /* Number of L blocks.                                */
// 	 int_t  *ilsum = Llu->ilsum; /* Starting position of each supernode in lsum.   */
	 
// 	 int_t bid;
// 	 int_t tmp;
// 	 int_t tid = threadIdx_x + threadIdx_y * blockDim_x; 
// 	 int_t ready = 0;
// 	 // int_t lock = 0;
// 	 const int block_size = blockDim_x*blockDim_y; /* number of threads per block*/
// 	 double zero = 0.0;
 
	 
// 	 double rC[THR_N][THR_M];
	 
// 	 gpuError_t error;
	 
// 	 bid= blockIdx_x;
// 	 int_t idx = threadIdx_x;  // thread's m dimension
// 	 int_t idy = threadIdx_y;  // thread's n dimension
// 	 int_t ni,mi;
	 
	 
// 	 int_t wrp;
// 	 int_t lne = threadIdx_x & 0x1f ;
// 	 // int_t ready = 0;
// 	 // int_t lock = 0;
// 	 const int warp_size = 32; /* number of threads per warp*/
// 	 wrp= threadIdx_x + blockIdx_x * blockDim_x;
// 	 wrp/=warp_size;	
// 	 const int wrp_loc = threadIdx_x / NWARP;
// 	 int starting_x = (tid / (NWARP * warp_size)) * NWARP;
//  #ifdef USESHARE1RHS		
// 	 if (idx < NWARP) { s_fmod[idx] = 0;}
// 	 for (i = idx; i < NWARP*MaxSUP; i+=block_size){s_lsum[i]=zero;}
//  #endif	
// 	 __syncthreads();
	 
	 
// 	 // printf("  Entering kernel:   %i %i %i %i %i %i %i %i\n", threadIdx_x, blockIdx_x, grid->npcol, nsupers,myrow,krow,bid,tid);
	 
	 
// 	 // rtemp_loc = (double*)malloc(maxsup*nrhs*Nbk*sizeof(double));
	 
// 	 if(wrp>=CEILING(nsupers, grid->npcol)){
// 	 return;
// 	 }else if(!Llu->Lrowind_bc_ptr[wrp]){
// 	 return;
// 	 }
	 
	 
 
// 	 lk=wrp;
// 	 iam = grid->iam;
// 	 mycol = MYCOL( iam, grid );
// 	 myrow = MYROW( iam, grid );
// 	 k = mycol+lk*grid->npcol;
// 	 knsupc = SuperSize( k );
// 	 lsub = Llu->Lrowind_bc_ptr[lk];
// 	 iam = grid->iam;
// 	 krow = PROW( k, grid );	
// 	 lusup = Llu->Lnzval_bc_ptr[lk];
// 	 lloc = Llu->Lindval_loc_bc_ptr[lk];
// 	 nsupr = lsub[1];
	 
// 	 if(myrow==krow){
// 		 nlb = lsub[0] - 1;
// 		 idx_n = 1;
// 		 idx_i = nlb+2;
// 		 idx_v = 2*nlb+3;
// 		 luptr_tmp = lloc[idx_v];
// 		 m = nsupr-knsupc;
// 	 }else{
// 		 nlb = lsub[0];
// 		 idx_n = 0;
// 		 idx_i = nlb;
// 		 idx_v = 2*nlb;
// 		 luptr_tmp = lloc[idx_v];
// 		 m = nsupr;
// 	 }	
	 
// 	 // printf("  Before kernel:   %i %i %i %i %i %i %i %i\n", threadIdx_x, blockIdx_x, grid->npcol, nsupers,myrow,krow,bid,tid);
	 
// 	 if(myrow==krow){   /* diagonal block performs trsm and forward the message*/
 
// 		 if(lne==0){  /*only the first thread in a warp handles the lock */
 
// 		 // printf("bk: %5d r: %5d %5d %5d\n",mycol+bid*grid->npcol,fmod[2*aln_i],myrow,krow);
// 		 // for (i=0 ; i<maxsup ; i++){
// 			 // rtemp_loc[i]=0.0;
// 		 // }	
		 
// 			 lib = LBi( k, grid ); /* Local block number, row-wise. */
// 			 do{
//  #ifdef USESHARE1RHS				
// 				 tmp=fmod[lib]+s_fmod[wrp_loc];
//  #else
// 				 tmp=fmod[lib];
//  #endif				
// 				 __threadfence();			
// 			 }while(tmp>0);
			 
// 		 }
// 		 __syncwarp();
		 
			 
// 			 lib = LBi( k, grid ); /* Local block number, row-wise. */
// 			 il = LSUM_BLK( lib );
// 			 ii = X_BLK( lib );
			 
// 			 RHS_ITERATE(j)
// 				 for (i = lne; i < knsupc; i+=warp_size)
//  #ifdef USESHARE1RHS					
// 					 x[i + ii + j*knsupc] += (lsum[i + il + j*knsupc ]+s_lsum[i+wrp_loc*MaxSUP]);
//  #else					
// 					 x[i + ii + j*knsupc] += (lsum[i + il + j*knsupc ]);
//  #endif					
// 			 // __syncwarp();
			 
			 
// 			 if(Llu->inv == 1){
			 
// 				 Linv = Llu->Linv_bc_ptr[lk];
					 
// 				 if(nrhs==1){
				 
// 					 for (i = lne; i < knsupc; i+=warp_size){					
// 						 temp1=zero;
// 						 for (l=0 ; l<knsupc ; l++){
// 							 temp1+=  Linv[l*knsupc+i]*x[ii+l];
// 						 }								
// 						 lsum[il+i]=temp1; //reuse lsum as temporary output as it's no longer accessed
// 					 }
// 					 // __syncwarp();					
						 
// 					 for (i = lne; i < knsupc; i+=warp_size){
// 						 x[i + ii] = lsum[il+i];
// 						 // printf("lk %5d %lf\n",lk,x[i + ii + j*knsupc]);
// 						 }					
// 					 // __syncwarp();		
						 
 
					 
// 					 // RHS_ITERATE(j){
					 
// 					 // for (i = lne; i < knsupc; i+=warp_size)
// 						 // rtemp_loc[i]=zero;					
// 					 // __syncwarp(); 
					 
									 
// 					 // gemv_device_dlsum_fmod(
// 						 // knsupc, knsupc, alpha,
// 						 // Linv, knsupc,
// 						 // &x[ii+j*knsupc], 1, beta,
// 						 // rtemp_loc, 1);											
						 
// 					 // __syncwarp(); 
// 					 // // printf("lne %5d knsupc %5d warp_size %5d\n",lne,knsupc,warp_size);
// 					 // for (i = lne; i < knsupc; i+=warp_size){
// 						 // x[i + ii + j*knsupc] = rtemp_loc[i];
// 						 // // printf("lk %5d %lf\n",lk,x[i + ii + j*knsupc]);
// 						 // }
// 					 // }	
// 					 // __syncwarp(); 	
					 
// 				 }else{
// 					 __syncwarp(); 	
// 					 for (int_t blx = 0; blx*BLK_M < knsupc; blx++){
// 						 for (int_t bly = 0; bly*BLK_N < nrhs; bly++){
// 							 gemm_device_dlsum_fmod(knsupc, nrhs, knsupc, blx, bly, 
// 							 Linv, knsupc, &x[ii], knsupc, rC,
// 							 alpha, beta);
// 								 #pragma unroll
// 							 for (ni = 0; ni < THR_N; ni++) {
// 								 int_t coord_dCn = bly*BLK_N + ni*DIM_Y + idy;
// 								 #pragma unroll
// 								 for (mi = 0; mi < THR_M; mi++) {
// 									 int_t coord_dCm = blx*BLK_M + mi*DIM_X + idx;
// 									 if (coord_dCm < knsupc && coord_dCn < nrhs) {
// 										 double &regC = rC[ni][mi];
// 										 lsum[coord_dCm + il + coord_dCn*knsupc ]=regC;  //reuse lsum as temporary output as it's no longer accessed
// 									 }//if (coord_dCm < knsupc && coord_dCn < nrhs)
// 								 }
// 							 }						
// 						 }
// 					 }
// 					 __syncwarp(); 	
 
// 					 RHS_ITERATE(j)
// 					 for (i = lne; i < knsupc; i+=warp_size)
// 						 x[i + ii + j*knsupc] = lsum[i + il + j*knsupc ];
// 					 __syncwarp(); 		
// 				 }//if(nrhs==1)
// 			 }
			 
			 
// 		 // __syncwarp();	
// 	 }else{   /* off-diagonal block forward the message*/
// 		 /* waiting for the x subvector and forward*/ 
// 	 }
	 
	   
// 	 if(nlb>0){
	 
// 			 lib = LBi( k, grid ); /* Local block number, row-wise. */
// 			 ii = X_BLK( lib );	
			 
// 			 // if(nrhs==1){
// 				 luptr_tmp1 = lloc[idx_v];
// 				 lb = 0;
// 				 nbrow=0;
// 				 lptr1_tmp = lloc[lb+idx_i];
// 				 lptr= lptr1_tmp+2;
// 				 nbrow1 = lsub[lptr1_tmp+1];
// 				 ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
// 				 rel = xsup[ik]; /* Global row index of block ik. */
// 				 lk = LBi( ik, grid ); /* Local block number, row-wise. */
// 				 iknsupc = SuperSize( ik );
// 				 il = LSUM_BLK( lk );			
				 
// 				 for (i = lne; i < m; i+=warp_size){
// 					 while(nbrow+lsub[lptr1_tmp+1]<=i){
// 						 lb++;
// 						 nbrow +=lsub[lptr1_tmp+1];
// 						 lptr1_tmp = lloc[lb+idx_i];
// 						 lptr= lptr1_tmp+2;
// 						 ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
// 						 rel = xsup[ik]; /* Global row index of block ik. */
// 						 lk = LBi( ik, grid ); /* Local block number, row-wise. */
// 						 iknsupc = SuperSize( ik );
// 						 il = LSUM_BLK( lk );				
// 					 }
					 
// 					 irow = lsub[lptr+i-nbrow] - rel; /* Relative row. */
// 					 RHS_ITERATE(j){
// 					 temp1=zero;
// 					 for (l=0 ; l<knsupc ; l++){
// 						 temp1+= lusup[luptr_tmp1+l*nsupr+i]*x[ii+j*knsupc+l];
// 					 }
//  #ifdef USESHARE1RHS					
// 					 if(lk<starting_x+NWARP){
// 						 temp=atomicAdd((double *)&s_lsum[irow+(lk-starting_x)*MaxSUP],-temp1);
// 					 }else{
// 						 temp=atomicAdd(&lsum[il+irow + j*iknsupc],-temp1);
// 					 }
//  #else
// 					 temp=atomicAdd(&lsum[il+irow + j*iknsupc],-temp1);
//  #endif							
// 					 }
			 
// 					 if(i==nbrow+lsub[lptr1_tmp+1]-1){
//  #ifdef USESHARE1RHS					
// 						 if(lk<starting_x+NWARP){
// 							 fmod_tmp=atomicSub((int *)&s_fmod[lk-starting_x],1);
// 						 }else{
// 							 fmod_tmp=atomicSub(&fmod[lk],1);
// 						 }
//  #else						
// 						 fmod_tmp=atomicSub(&fmod[lk],1);
//  #endif							
// 						 // __threadfence();
// 					 }
// 				 }
// 				 // __syncwarp();
// 			 // }//if(nrhs==1)
		 
			 
// 			 // if(tid==0){
// 			 // for (lb = tid; lb < nlb; lb+=warp_size){
// 					 // lptr1_tmp = lloc[lb+idx_i];
// 					 // ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
// 					 // lk = LBi( ik, grid ); /* Local block number, row-wise. */
// 					 // fmod_tmp=atomicSub(&fmod[lk*aln_i],1);
// 					 // // printf("k: %5d r: %5d\n",mycol+bid*grid->npcol,fmod[2*aln_i]);
// 			 // }
// 			 // }
// 			 // __syncwarp();
// 		 // } /*if tid<Nchunk*/
// 	 } /* if nlb>0*/		
		 
	 
//  } /* dlsum_fmod_inv_gpu_1rhs */
 
 
 
 
 
 // /************************************************************************/
 // /*! \brief
  // *
  // * <pre>
  // * Purpose
  // * =======
  // *   Perform local block modifications: lsum[i] -= L_i,k * X[k].
  // * </pre>
  // */
 // __global__ void dlsum_fmod_inv_gpu_1rhs
 // /************************************************************************/
 // (
  // double *lsum,    /* Sum of local modifications.                        */
  // double *x,       /* X array (local)                                    */
  // double *rtemp,   /* Result of full matrix-vector multiply.             */
  // int   nrhs,      /* Number of right-hand sides.                        */
  // int   maxsup,      /* Max supernode size.                        */
  // int_t   nsupers,      /* Number of total supernodes.                        */
  // int_t *fmod,     /* Modification count for L-solve.                    */
  // int_t *xsup,
  // gridinfo_t *grid,
  // LocalLU_t *Llu
 // )
 // {
	 // double alpha = 1.0, beta = 0.0,malpha=-1.0;
	 // double *lusup, *lusup1;
	 // double *dest;
	 // double *Linv;/* Inverse of diagonal block */
	 // int    iam, iknsupc, myrow, mycol, krow, nbrow, nbrow1, nbrow_ref, nsupr, nsupr1, p, pi, idx_r,m;
	 // int_t  k,i, l,ii,jj, ik, il, ikcol, irow, j, lb, lk, rel, lib,lready;
	 // int_t  *lsub, *lsub1, nlb1, lptr1, luptr1,*lloc;
	 // int_t  luptr_tmp,luptr_tmp1,lptr1_tmp,maxrecvsz, idx_i, idx_v,idx_n,  idx_l, fmod_tmp, lbstart,lbend,nn,Nchunk,nlb_loc,remainder;
	 // int thread_id1;
	 // flops_t ops_loc=0.0;
	 // MPI_Status status;
	 // int test_flag;
	 // yes_no_t done;
	 // C_Tree  *LBtree_ptr = Llu->LBtree_ptr;
	 // C_Tree  *LRtree_ptr = Llu->LRtree_ptr;
	 // int_t* idx_lsum,idx_lsum1;
	 // const int Nbk=1;
	 // // __shared__ double rtemp_loc[128]; 
	 // double temp,temp1;
	 // int_t ldalsum;
	 // int_t nleaf_send_tmp;
	 // int_t lptr;      /* Starting position in lsub[*].                      */
	 // int_t luptr;     /* Starting position in lusup[*].                     */
	 // int_t iword = sizeof(int_t);
	 // int_t dword = sizeof (double);
	 // int_t aln_d,aln_i;
	 // aln_d = 1;//ceil(CACHELINE/(double)dword);
	 // aln_i = 1;//ceil(CACHELINE/(double)iword);
	 // int   knsupc;    /* Size of supernode k.                               */
	 // int_t nlb;       /* Number of L blocks.                                */
	 // int_t  *ilsum = Llu->ilsum; /* Starting position of each supernode in lsum.   */
	 
	 // int_t bid;
	 // int_t tmp;
	 // int_t tid = threadIdx_x + threadIdx_y * blockDim_x; 
	 // int_t ready = 0;
	 // // int_t lock = 0;
	 // const int block_size = blockDim_x*blockDim_y; /* number of threads per block*/
	 // double zero = 0.0;
 
 
	 // double rC[THR_N][THR_M];
	 
	 // gpuError_t error;
	 
	 // bid= blockIdx_x;
	 // int_t idx = threadIdx_x;  // thread's m dimension
	 // int_t idy = threadIdx_y;  // thread's n dimension
	 // int_t ni,mi;
	 
	 
	 // int_t wrp;
	 // int_t lne = threadIdx_x & 0x1f ;
	 // // int_t ready = 0;
	 // // int_t lock = 0;
	 // const int warp_size = 32; /* number of threads per warp*/
	 // wrp= threadIdx_x + blockIdx_x * blockDim_x;
	 // wrp/=warp_size;	
	 
	 
	 
	 // // printf("  Entering kernel:   %i %i %i %i %i %i %i %i\n", threadIdx_x, blockIdx_x, grid->npcol, nsupers,myrow,krow,bid,tid);
	 
	 
	 // // rtemp_loc = (double*)malloc(maxsup*nrhs*Nbk*sizeof(double));
	 
	 // if(wrp>=CEILING(nsupers, grid->npcol)){
	 // return;
	 // }else if(!Llu->Lrowind_bc_ptr[wrp]){
	 // return;
	 // }
	 
	 
 
	 // lk=wrp;
	 // iam = grid->iam;
	 // mycol = MYCOL( iam, grid );
	 // myrow = MYROW( iam, grid );
	 // k = mycol+lk*grid->npcol;
	 // knsupc = SuperSize( k );
	 // lsub = Llu->Lrowind_bc_ptr[lk];
	 // iam = grid->iam;
	 // krow = PROW( k, grid );	
	 // lusup = Llu->Lnzval_bc_ptr[lk];
	 // lloc = Llu->Lindval_loc_bc_ptr[lk];
	 // nsupr = lsub[1];
	 
	 // if(myrow==krow){
		 // nlb = lsub[0] - 1;
		 // idx_n = 1;
		 // idx_i = nlb+2;
		 // idx_v = 2*nlb+3;
		 // luptr_tmp = lloc[idx_v];
		 // m = nsupr-knsupc;
	 // }else{
		 // nlb = lsub[0];
		 // idx_n = 0;
		 // idx_i = nlb;
		 // idx_v = 2*nlb;
		 // luptr_tmp = lloc[idx_v];
		 // m = nsupr;
	 // }	
	 
	 // // printf("  Before kernel:   %i %i %i %i %i %i %i %i\n", threadIdx_x, blockIdx_x, grid->npcol, nsupers,myrow,krow,bid,tid);
	 
	 // if(myrow==krow){   /* diagonal block performs trsm and forward the message*/
 
		 // if(lne==0){  /*only the first thread in a warp handles the lock */
 
		 // // printf("bk: %5d r: %5d %5d %5d\n",mycol+bid*grid->npcol,fmod[2*aln_i],myrow,krow);
		 // // for (i=0 ; i<maxsup ; i++){
			 // // rtemp_loc[i]=0.0;
		 // // }	
		 
			 // lib = LBi( k, grid ); /* Local block number, row-wise. */
			 // do{
				 // tmp=fmod[lib*aln_i];
				 // __threadfence();			
			 // }while(tmp>0);
			 
		 // }
		 // __syncwarp();
		 
			 
		 // lib = LBi( k, grid ); /* Local block number, row-wise. */
		 // il = LSUM_BLK( lib );
		 // ii = X_BLK( lib );
	 
		 // for (i = lne; i < knsupc; i+=warp_size)
			 // x[i + ii ] += lsum[i + il ];
		 // // __syncwarp();
		 
		 
		 // if(Llu->inv == 1){
			 // Linv = Llu->Linv_bc_ptr[lk];
			 // for (i = lne; i < knsupc; i+=warp_size){					
				 // temp1=zero;
				 // for (l=0 ; l<knsupc ; l++){
					 // temp1+=  Linv[l*knsupc+i]*x[ii+l];
				 // }								
				 // lsum[il+i]=temp1; //reuse lsum as temporary output as it's no longer accessed
			 // }
			 // // __syncwarp();
							 
			 // for (i = lne; i < knsupc; i+=warp_size){
				 // x[i + ii] = lsum[il+i];
				 // // printf("lk %5d %lf\n",lk,x[i + ii + j*knsupc]);
				 // }					
			 // // __syncwarp();		
 
		 // }
		 // // __syncwarp();	
	 // }else{   /* off-diagonal block forward the message*/
		 // /* waiting for the x subvector and forward*/ 
	 // }
	 
	   
	 // if(nlb>0){
			 // if(nrhs==1){
			 // lib = LBi( k, grid ); /* Local block number, row-wise. */
			 // ii = X_BLK( lib );	
			 
			 // luptr_tmp1 = lloc[idx_v];
			 // lb = 0;
			 // nbrow=0;
			 // lptr1_tmp = lloc[lb+idx_i];
			 // lptr= lptr1_tmp+2;
			 // nbrow1 = lsub[lptr1_tmp+1];
			 // ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
			 // rel = xsup[ik]; /* Global row index of block ik. */
			 // lk = LBi( ik, grid ); /* Local block number, row-wise. */
			 // iknsupc = SuperSize( ik );
			 // il = LSUM_BLK( lk );			
			 // for (i = lne; i < m; i+=warp_size){
				 // while(nbrow+lsub[lptr1_tmp+1]<=i){
					 // lb++;
					 // nbrow +=lsub[lptr1_tmp+1];
					 // lptr1_tmp = lloc[lb+idx_i];
					 // lptr= lptr1_tmp+2;
					 // ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
					 // rel = xsup[ik]; /* Global row index of block ik. */
					 // lk = LBi( ik, grid ); /* Local block number, row-wise. */
					 // iknsupc = SuperSize( ik );
					 // il = LSUM_BLK( lk );				
				 // }
				 
				 // irow = lsub[lptr+i-nbrow] - rel; /* Relative row. */
				 // RHS_ITERATE(j){
				 // temp1=zero;
				 // for (l=0 ; l<knsupc ; l++){
					 // temp1+= lusup[luptr_tmp1+l*nsupr+i]*x[ii+j*knsupc+l];
				 // }
	 
				 // temp=atomicAdd(&lsum[il+irow + j*iknsupc],-temp1);
				 // }
				 // if(i==nbrow+lsub[lptr1_tmp+1]-1){
					 // fmod_tmp=atomicSub(&fmod[lk*aln_i],1);
					 // // __threadfence();
				 // }
			 // }
			 // }
			 // // __syncwarp();
 
			 // // if(tid==0){
			 // // for (lb = tid; lb < nlb; lb+=warp_size){
					 // // lptr1_tmp = lloc[lb+idx_i];
					 // // ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
					 // // lk = LBi( ik, grid ); /* Local block number, row-wise. */
					 // // fmod_tmp=atomicSub(&fmod[lk*aln_i],1);
					 // // // printf("k: %5d r: %5d\n",mycol+bid*grid->npcol,fmod[2*aln_i]);
			 // // }
			 // // }
			 // // __syncwarp();
		 // // } /*if tid<Nchunk*/
	 // } /* if nlb>0*/		
		 
	 
 // } /* dlsum_fmod_inv_gpu_1rhs */

__inline__ __device__
int warpReduceSum(int val) {
  for (int offset = warpSize/2; offset > 0; offset /= 2)
    //val += __shfl_down_sync(0xffffffff,val, offset,warpSize);
    val += __shfl_down_sync(0xffffffff, val, offset, warpSize);
    //__shfl_down_sync(unsigned mask, T var, unsigned int delta, int width=warpSize);
  return val;
}

__inline__ __device__
int warpAllReduceSum(int val) {
  for (int mask = warpSize/2; mask > 0; mask /= 2)
    val += __shfl_xor_sync(0xffffffff,val, mask,warpSize);
  return val;
}

__inline__ __device__
int blockReduceSum(int val, int bid, int tid, int mype) {

  static __shared__ int shared[32]; // Shared mem for 32 partial sums
  double sz=32.0;
  int lane = tid % warpSize;
  int wid = tid>>(int)log2(sz);
  val = warpReduceSum(val);     // Each warp performs partial reduction

  if (lane==0) shared[wid]=val; // Write reduced value to shared memory
  __syncthreads();              // Wait for all partial reductions

  //read from shared memory only if that warp existed
  val = (tid < (blockDim.x * blockDim.y) / warpSize) ? shared[lane] : 0;

  if (wid==0) val = warpReduceSum(val); //Final reduce within first warp

  return val;
}


__inline__ __device__ int warpReduceMin(int val)
{
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        int tmpVal = __shfl_down_sync(0xffffffff,val, offset, warpSize);
        if (tmpVal < val)  val = tmpVal;
    }
    return val;
}

__inline__ __device__  int blockReduceMin(int val,int bid, int tid, int mype)
{

    static __shared__ int shared[32]; // Shared mem for 32 partial mins
    double sz=32.0;
    int lane = tid % warpSize;
    int wid = tid>>(int)log2(sz);

    warpReduceMin(val);     // Each warp performs partial reduction

    if (lane == 0) shared[wid] = val; // Write reduced value to shared memory

    __syncthreads();              // Wait for all partial reductions

    //read from shared memory only if that warp existed
    val = (tid < (blockDim.x * blockDim.y) / warpSize) ? shared[lane] : INT_MAX;

    if (wid == 0)  warpReduceMin(val); //Final reduce within first warp
    return val;
}

 /************************************************************************/
 /*! \brief
  *
  * <pre>
  * Purpose
  * =======
  *   Perform local block modifications: lsum[i] -= L_i,k * X[k].
  * </pre>
  */
__global__ void dlsum_fmod_inv_gpu_mrhs_nvshmem
/************************************************************************/
(
        int_t nbcol_loc,
        int_t nblock_ex,
        double *lsum,    /* Sum of local modifications.                        */
        double *x,       /* X array (local)                                    */
        int   nrhs,      /* Number of right-hand sides.                        */
        int   maxsup,      /* Max supernode size.                        */
        int_t   nsupers,      /* Number of total supernodes.                        */
        int_t *fmod,     /* Modification count for L-solve.                    */
        C_Tree  *LBtree_ptr,
        C_Tree  *LRtree_ptr,
        int_t *ilsum,
        int_t *Lrowind_bc_dat,
        long int *Lrowind_bc_offset,
        double *Lnzval_bc_dat,
        long int *Lnzval_bc_offset,
        double *Linv_bc_dat,
        long int *Linv_bc_offset,
        int_t *Lindval_loc_bc_dat,
        long int *Lindval_loc_bc_offset,
        int_t *xsup,
        gridinfo_t *grid,
        int_t maxrecvsz,
        int mype,
        volatile int* flag_bc_q,
        volatile int* flag_rd_q,
        double* ready_x,
        double* ready_lsum,
        int* my_flag_bc,
        int* my_flag_rd,
        int totalth,
        int* d_launch_flag,
        int* d_nfrecv,
        volatile int* d_status,
        volatile int* d_statusmod
)
{
    double alpha = 1.0, beta = 0.0,malpha=-1.0;
    double *lusup, *lusup1;
    double *dest;
    double *Linv;/* Inverse of diagonal block */
    int    iam, iknsupc, myrow, mycol, krow, nbrow, nbrow1, nbrow_ref, nsupr, nsupr1, p, pi, idx_r,m;
    int_t  k,i, l,ii,jj, ik, il, ikcol, irow, j, lb, lk, rel, lib,lready;
    int_t  *lsub, *lsub1, nlb1, lptr1, luptr1,*lloc;
    int_t  luptr_tmp,luptr_tmp1,lptr1_tmp, idx_i, idx_v,idx_n,  idx_l, fmod_tmp, lbstart,lbend,nn,Nchunk,nlb_loc,remainder;
    int thread_id1;
    flops_t ops_loc=0.0;
    MPI_Status status;
    int test_flag;
    yes_no_t done;
    int_t* idx_lsum,idx_lsum1;
    const int Nbk=1;
    __shared__ double rtemp_loc[128];
    double temp,temp1;
    int_t ldalsum;
    int_t nleaf_send_tmp;
    int_t lptr;      /* Starting position in lsub[*].                      */
    int_t luptr;     /* Starting position in lusup[*].                     */
    int_t iword = sizeof(int_t);
    int_t dword = sizeof (double);
    int_t aln_d,aln_i;
    aln_d = 1;//ceil(CACHELINE/(double)dword);
    aln_i = 1;//ceil(CACHELINE/(double)iword);
    int   knsupc;    /* Size of supernode k.                               */
    int_t nlb;       /* Number of L blocks.                                */

    int_t bid;
    int_t tmp;
    int_t tid = threadIdx_x + threadIdx_y * blockDim_x;
    int_t ready = 0;
    // int_t lock = 0;
    const int block_size = blockDim_x*blockDim_y; /* number of threads per warp*/
    double zero = 0.0;


    double rC[THR_N][THR_M];

    gpuError_t error;

    bid= blockIdx_x;
    int_t idx = threadIdx_x;  // thread's m dimension
    int_t idy = threadIdx_y;  // thread's n dimension
    int_t ni,mi;
    int cnt;
    yes_no_t test;

    //if (  (mype==0) && (tid==0) && (bid>150) ) printf(" (%d,%d) Entering kernel\n", mype,bid);


    // rtemp_loc = (double*)malloc(maxsup*nrhs*Nbk*sizeof(double));


    // the first nbcol_loc handles all computations and broadcast communication
    if(bid<nbcol_loc){


        if(Lrowind_bc_offset[bid]==-1){
            return;
        }


        lk=bid;
        iam = grid->iam;
        mycol = MYCOL( iam, grid );
        myrow = MYROW( iam, grid );
        k = mycol+lk*grid->npcol;
        knsupc = SuperSize( k );
        lsub = &Lrowind_bc_dat[Lrowind_bc_offset[lk]];
        iam = grid->iam;
        krow = PROW( k, grid );
        lusup = &Lnzval_bc_dat[Lnzval_bc_offset[lk]];
        lloc = &Lindval_loc_bc_dat[Lindval_loc_bc_offset[lk]];
        nsupr = lsub[1];
        int get_offset, get_msgsize, get_rank, gc, gr, tmp_id,recv_offset=0;
        gc=mycol+lk*grid->npcol;
        if( gc>=nsupers) return;

        if(myrow==krow){
            nlb = lsub[0] - 1;
            idx_n = 1;
            idx_i = nlb+2;
            idx_v = 2*nlb+3;
            luptr_tmp = lloc[idx_v];
            m = nsupr-knsupc;
        }else{
            nlb = lsub[0];
            idx_n = 0;
            idx_i = nlb;
            idx_v = 2*nlb;
            luptr_tmp = lloc[idx_v];
            m = nsupr;
        }

        // printf("  Before kernel:   %i %i %i %i %i %i %i %i\n", threadIdx_x, blockIdx_x, grid->npcol, nsupers,myrow,krow,bid,tid);
        if(myrow==krow){   /* diagonal block performs trsm and forward the message*/

            if(tid==0){  /*only the first thread in a block handles the lock */

                // printf("bk: %5d r: %5d %5d %5d\n",mycol+bid*grid->npcol,fmod[2*aln_i],myrow,krow);
                // for (i=0 ; i<maxsup ; i++){
                // rtemp_loc[i]=0.0;
                // }

                lib = LBi( k, grid ); /* Local block number, row-wise. */
                do{
                    tmp=fmod[lib*aln_i];
                    __threadfence();
                }while(tmp>0);

            }
            __syncthreads();


            lib = LBi( k, grid ); /* Local block number, row-wise. */
            il = LSUM_BLK( lib );
            ii = X_BLK( lib );

            RHS_ITERATE(j)
                for (i = tid; i < knsupc; i+=block_size) {
                    //atomicAdd(&ready_x[0],lsum[i + il + j * knsupc]);
                    x[i + ii + j * knsupc] += lsum[i + il + j * knsupc];
                }
            __syncthreads();
            //if(tid==0) printf("(%d,%d,%d),CHECKING k=%d,gc=%d,checksum=%lf\n",mype,bid,tid,k,gc,ready_x[0]);


            //  if(Llu->inv == 1){

            Linv = &Linv_bc_dat[Linv_bc_offset[lk]];

            if(nrhs==1){

                for (i = tid; i < knsupc; i+=block_size){
                    temp1=zero;
                    for (l=0 ; l<knsupc ; l++){
                        temp1+=  Linv[l*knsupc+i]*x[ii+l];
                    }
                    lsum[il+i]=temp1; //reuse lsum as temporary output as it's no longer accessed
                }
                __syncthreads();

                for (i = tid; i < knsupc; i+=block_size){
                    x[i + ii] = lsum[il+i];
                    // printf("lk %5d %lf\n",lk,x[i + ii + j*knsupc]);
                }
                __syncthreads();



                // RHS_ITERATE(j){

                // for (i = tid; i < knsupc; i+=block_size)
                // rtemp_loc[i]=zero;
                // __syncthreads();


                // gemv_device_dlsum_fmod(
                // knsupc, knsupc, alpha,
                // Linv, knsupc,
                // &x[ii+j*knsupc], 1, beta,
                // rtemp_loc, 1);

                // __syncthreads();
                // // printf("tid %5d knsupc %5d block_size %5d\n",tid,knsupc,block_size);
                // for (i = tid; i < knsupc; i+=block_size){
                // x[i + ii + j*knsupc] = rtemp_loc[i];
                // // printf("lk %5d %lf\n",lk,x[i + ii + j*knsupc]);
                // }
                // }
                // __syncthreads();

            }else{
                __syncthreads();
                for (int_t blx = 0; blx*BLK_M < knsupc; blx++){
                    for (int_t bly = 0; bly*BLK_N < nrhs; bly++){
                        gemm_device_dlsum_fmod(knsupc, nrhs, knsupc, blx, bly,
                                               Linv, knsupc, &x[ii], knsupc, rC,
                                               alpha, beta);
#pragma unroll
                        for (ni = 0; ni < THR_N; ni++) {
                            int_t coord_dCn = bly*BLK_N + ni*DIM_Y + idy;
#pragma unroll
                            for (mi = 0; mi < THR_M; mi++) {
                                int_t coord_dCm = blx*BLK_M + mi*DIM_X + idx;
                                if (coord_dCm < knsupc && coord_dCn < nrhs) {
                                    double &regC = rC[ni][mi];
                                    lsum[coord_dCm + il + coord_dCn*knsupc ]=regC;  //reuse lsum as temporary output as it's no longer accessed
                                }//if (coord_dCm < knsupc && coord_dCn < nrhs)
                            }
                        }
                    }
                }
                __syncthreads();

                RHS_ITERATE(j)
                    for (i = tid; i < knsupc; i+=block_size)
                        x[i + ii + j*knsupc] = lsum[i + il + j*knsupc ];
                __syncthreads();
            }//if(nrhs==1)
            //  }

            RHS_ITERATE(j)
                for (i = tid; i < knsupc; i+=block_size)
                    ready_x[i + maxrecvsz*lk + j*knsupc ] = x[i + ii + j*knsupc];

            __syncthreads();
        }else{   /* off-diagonal block forward the message*/
            /* waiting for the x subvector and forward*/
              //YL: only the first thread in a block spin-waits for the coming x subvector message using NVSHMEM, put the message into ready_x[maxrecvsz*lk]
            volatile int msg_recv=0;
            if(tid==0){
                do{
                    msg_recv=flag_bc_q[gc];
                    //msg_recv=d_status[gc];
                    //msg_recv=flag_bc_q[gc];
                    __threadfence();
                }while(msg_recv!=1);
                //printf("(%d,%d,%d,%d) in compute kernel, I have msg=%d,sz=%d.ofset=%d\n",mype,bid,tid,gc,msg_recv,LBtree_ptr[lk].msgSize_*nrhs+XK_H,maxrecvsz*lk);
                //double sum=0;
                //for (int myi=0;myi<LBtree_ptr[lk].msgSize_*nrhs+XK_H;myi++){
                //    sum+=ready_x[maxrecvsz*lk+myi];
                //}
                //printf("(%d,%d,%d,%d), recv, sum=%lf,%lf\n",mype,bid,tid,gc,sum,sum);
            }
            __syncthreads();
            //for(int i=0;i<LBtree_ptr[lk].msgSize_*nrhs+XK_H;i++){
            //    ready_x[maxrecvsz*lk+i]=ready_x[maxrecvsz*gc+i];
            //}
            //__syncthreads();

        }
        __syncthreads();

        //YL: only the first thread in a block forwards the x subvector using NVSHMEM
        cnt=LBtree_ptr[lk].destCnt_;
        //if (tid==0) printf("(%d) good1 %5d%5d\n",mype,lk,cnt);
        if(cnt>0){
            //cnt=LBtree_ptr[lk].msgSize_;
// send data in cuda kernel
            my_flag_bc[gc*RDMA_FLAG_SIZE]=lk;
            my_flag_bc[gc*RDMA_FLAG_SIZE+1]=gc;
            my_flag_bc[gc*RDMA_FLAG_SIZE+2]=maxrecvsz*lk;
            //my_flag_bc[gc*RDMA_FLAG_SIZE+2]=maxrecvsz*lk;
            my_flag_bc[gc*RDMA_FLAG_SIZE+3]=LBtree_ptr[lk].msgSize_*nrhs+XK_H;
            C_BcTree_forwardMessageSimple_Device(&LBtree_ptr[lk], (int*)flag_bc_q, &my_flag_bc[gc*RDMA_FLAG_SIZE],mype,tid,&ready_x[maxrecvsz*lk]);
            //C_BcTree_forwardMessageSimple_Device(&LBtree_ptr[lk],&ready_x[maxrecvsz*lk],cnt*nrhs+XK_H);
            __syncthreads();
// send data in cuda kernel
// send data in nvshmem kernel

        }
        //}
        int keep_lk=lk;

        if(nlb>0){

            lib = LBi( k, grid ); /* Local block number, row-wise. */
            ii = X_BLK( lib );

            if(nrhs==1){
                luptr_tmp1 = lloc[idx_v];
                lb = 0;
                nbrow=0;
                lptr1_tmp = lloc[lb+idx_i];
                lptr= lptr1_tmp+2;
                nbrow1 = lsub[lptr1_tmp+1];
                ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
                rel = xsup[ik]; /* Global row index of block ik. */
                lk = LBi( ik, grid ); /* Local block number, row-wise. */
                iknsupc = SuperSize( ik );
                il = LSUM_BLK( lk );

                for (i = tid; i < m; i+=block_size){
                    while(nbrow+lsub[lptr1_tmp+1]<=i){
                        lb++;
                        nbrow +=lsub[lptr1_tmp+1];
                        lptr1_tmp = lloc[lb+idx_i];
                        lptr= lptr1_tmp+2;
                        ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
                        rel = xsup[ik]; /* Global row index of block ik. */
                        lk = LBi( ik, grid ); /* Local block number, row-wise. */
                        iknsupc = SuperSize( ik );
                        il = LSUM_BLK( lk );
                    }

                    irow = lsub[lptr+i-nbrow] - rel; /* Relative row. */
                    RHS_ITERATE(j){
                        temp1=zero;
                        for (l=0 ; l<knsupc ; l++){
                            temp1+= lusup[luptr_tmp1+l*nsupr+i]*ready_x[l + maxrecvsz*keep_lk + j*knsupc];
                        }

                        temp=atomicAdd(&lsum[il+irow + j*iknsupc],-temp1);
                        //printf("(%d,%d,%d),lsum[%d]=%f\n",mype,bid,tid,il+irow + j*iknsupc,lsum[il+irow + j*iknsupc]);
                    }

                    //  irow = lsub[lptr+i-nbrow] - rel; /* Relative row. */
                    //  if(i==nbrow+lsub[lptr1_tmp+1]-1){
                    // 	 fmod_tmp=atomicSub(&fmod[lk*aln_i],1);
                    // 	 // __threadfence();
                    //  }


                }
                __syncthreads();

                luptr_tmp1 = lloc[idx_v];
                lb = 0;
                nbrow=0;
                lptr1_tmp = lloc[lb+idx_i];
                lptr= lptr1_tmp+2;
                nbrow1 = lsub[lptr1_tmp+1];
                ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
                rel = xsup[ik]; /* Global row index of block ik. */
                lk = LBi( ik, grid ); /* Local block number, row-wise. */
                iknsupc = SuperSize( ik );
                il = LSUM_BLK( lk );

                for (i = tid; i < m; i+=block_size){
                    while(nbrow+lsub[lptr1_tmp+1]<=i){
                        lb++;
                        nbrow +=lsub[lptr1_tmp+1];
                        lptr1_tmp = lloc[lb+idx_i];
                        lptr= lptr1_tmp+2;
                        ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
                        rel = xsup[ik]; /* Global row index of block ik. */
                        lk = LBi( ik, grid ); /* Local block number, row-wise. */
                        iknsupc = SuperSize( ik );
                        il = LSUM_BLK( lk );
                    }

                    irow = lsub[lptr+i-nbrow] - rel; /* Relative row. */
                    if(i==nbrow+lsub[lptr1_tmp+1]-1){
                        fmod_tmp=atomicSub(&fmod[lk*aln_i],1);
                        // __threadfence();
                    }
                }
                __syncthreads();


            }else {
                for (lb = 0; lb < nlb; lb++){
                    luptr_tmp1 = lloc[lb+idx_v];

                    // nbrow=0;
                    // lptr1_tmp = lloc[lb+idx_i];
                    // nbrow += lsub[lptr1_tmp+1];


                    lib = LBi( k, grid ); /* Local block number, row-wise. */
                    ii = X_BLK( lib );

                    lptr1_tmp = lloc[lb+idx_i];
                    lptr= lptr1_tmp+2;
                    nbrow1 = lsub[lptr1_tmp+1];
                    ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
                    rel = xsup[ik]; /* Global row index of block ik. */

                    lk = LBi( ik, grid ); /* Local block number, row-wise. */

                    iknsupc = SuperSize( ik );
                    il = LSUM_BLK( lk );


                    // if(nrhs==1){

                    // for (i = tid; i < nbrow1; i+=block_size)
                    // rtemp_loc[i]=zero;
                    // __syncthreads();


                    // gemv_device_dlsum_fmod(
                    // nbrow1, knsupc, alpha,
                    // &lusup[luptr_tmp1], nsupr,
                    // &x[ii], 1, beta,
                    // rtemp_loc, 1);

                    // __syncthreads();
                    // for (i = tid; i < nbrow1; i+=block_size){
                    // irow = lsub[lptr+i] - rel; /* Relative row. */
                    // temp=atomicAdd(&lsum[il+irow],-rtemp_loc[i]);
                    // }
                    // }else{

                    for (int_t blx = 0; blx*BLK_M < nbrow1; blx++){
                        for (int_t bly = 0; bly*BLK_N < nrhs; bly++){
                            gemm_device_dlsum_fmod(nbrow1, nrhs, knsupc, blx, bly,
                                                   &lusup[luptr_tmp1], nsupr, &ready_x[maxrecvsz*keep_lk], knsupc, rC,
                                                   alpha, beta);
#pragma unroll
                            for (ni = 0; ni < THR_N; ni++) {
                                int_t coord_dCn = bly*BLK_N + ni*DIM_Y + idy;
#pragma unroll
                                for (mi = 0; mi < THR_M; mi++) {
                                    int_t coord_dCm = blx*BLK_M + mi*DIM_X + idx;
                                    if (coord_dCm < nbrow1 && coord_dCn < nrhs) {
                                        irow = lsub[lptr+coord_dCm] - rel; /* Relative row. */
                                        double &regC = rC[ni][mi];
                                        temp=atomicAdd(&lsum[il+irow + coord_dCn*iknsupc],-regC);
                                    }
                                }
                            }
                        }
                    }
                    // }//if(nrhs==1)

                    if(tid==0)fmod_tmp=atomicSub(&fmod[lk*aln_i],1);



                }

            }//if(nrhs==1)


            // if(tid==0){
            // for (lb = tid; lb < nlb; lb+=block_size){
            // lptr1_tmp = lloc[lb+idx_i];
            // ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
            // lk = LBi( ik, grid ); /* Local block number, row-wise. */
            // fmod_tmp=atomicSub(&fmod[lk*aln_i],1);
            // // printf("k: %5d r: %5d\n",mycol+bid*grid->npcol,fmod[2*aln_i]);
            // }
            // }
            __syncthreads();
            // } /*if tid<Nchunk*/
        } /* if nlb>0*/


    }else if(bid<nbcol_loc+nblock_ex) {  //the next nblock_ex blocks handle all reduction communication
        //printf("HERE-(%d,%d,%d)\n", mype, bid, tid);
        int_t bid1 = bid - nbcol_loc;
        if (tid%32!=0) return;
        lib = bid1 * (block_size/32)+ tid/32; // the local numbering of my block row
        if (lib >= CEILING(nsupers, grid->nprow)) return;
        if (LRtree_ptr[lib].empty_ == YES) return;

        iam = grid->iam;
        mycol = MYCOL(iam, grid);
        myrow = MYROW(iam, grid);

        //lib = bid1 * block_size + tid; // the local numbering of my block row
        k = myrow + lib * grid->nprow; // global block row
        knsupc = SuperSize(k);
        il = LSUM_BLK(lib);
        //printf("HERE2-(%d,%d,%d),lib=%d,k=%d\n", mype, bid, tid, lib, k);

        cnt = LRtree_ptr[lib].destCnt_;

        volatile int msg=0;
        volatile int msg1=0;
        int orz=0;
        if (cnt > 0) {
            //printf("WAIT-(%d,%d,%d),lib=%d,k=%d,cnt=%d,msg=%d,sigofset=%d,%d\n", mype, bid, tid, lib, k, cnt, msg, k * 2,
            //       k * 2 + 1);
            do {
                //if ((mype == 0)) {
                //    printf("CHECK-%d,-(%d,%d,%d),flag=%d,%d\n", orz,mype, bid, tid, flag_rd_q[k * 2], flag_rd_q[k * 2 + 1]);
                //    printf("CHECK-%d,-(%d,%d,%d),status=%d,%d\n", orz,mype, bid, tid, d_statusmod[k * 2], d_statusmod[k * 2 + 1]);
                //}
                msg = d_statusmod[k*2]+d_statusmod[k*2+1];
                msg1 = flag_rd_q[k * 2] + flag_rd_q[k * 2 + 1];
                //msg=flag_rd_q[k*2+1];
                __threadfence();
                //if ((mype == 0)) {
                //    printf("CHECK2-%d,-(%d,%d,%d),flag=%d,%d\n", orz,mype, bid, tid, flag_rd_q[k * 2], flag_rd_q[k * 2 + 1]);
                //    printf("CHECK2-%d,-(%d,%d,%d),status=%d,%d\n", orz,mype, bid, tid, d_statusmod[k * 2], d_statusmod[k * 2 + 1]);
                //}
                //orz+=1;
            } while ((msg!=cnt) && (msg1!=cnt));
            //} while (msg1!=cnt);
        //} while (orz<10);
            printf("MSG-(%d,%d,%d),lib=%d,k=%d,cnt=%d\n", mype, bid, tid, lib, k, cnt);

            double tmp_sum=0;
            ////YL: wait for the one or two coming messages to complete using NVSHMEM, the received data is in ready_lsum[maxrecvsz*lib*2]
            if (cnt==2) {
                for (ii = 0; ii < cnt; ++ii) {
                    tmp_sum=0;
                    RHS_ITERATE(j) {
                        for (i = 0; i < knsupc; ++i) {
                            //temp=atomicAdd(&lsum[il+i + j*knsupc], ready_lsum[maxrecvsz*lib*2+ii*maxrecvsz + i + j*knsupc]  );
                            temp = atomicAdd(&lsum[il + i + j * knsupc],
                                             ready_lsum[maxrecvsz * k * 2 + ii * maxrecvsz + i + j * knsupc]);
                            tmp_sum += ready_lsum[maxrecvsz * lib * 2 + ii * maxrecvsz + i + j * knsupc];
                            //printf("data2-(%d,%d,%d),lib=%d,k=%d,ii=%d,ready_lsum[%d]=%f\n", mype, bid, tid,
                            //       lib, k, ii,
                            //       maxrecvsz * lib * 2 + ii * maxrecvsz + i + j * knsupc,
                            //       ready_lsum[maxrecvsz * lib * 2 + ii * maxrecvsz + i + j * knsupc]);
                        }

                        printf("sum2-(%d,%d,%d),lib=%d,k=%d,sum=%f\n", mype, bid, tid, lib, k, tmp_sum);//////////
                        fmod_tmp = atomicSub(&fmod[lib * aln_i], 1);
                    }
                }
            }
            if (cnt==1){
                if (flag_rd_q[k * 2+1]==1) ii=1;
                RHS_ITERATE(j) {
                    for (i = 0; i < knsupc; ++i) {
                        //temp=atomicAdd(&lsum[il+i + j*knsupc], ready_lsum[maxrecvsz*lib*2+ii*maxrecvsz + i + j*knsupc]  );
                        temp = atomicAdd(&lsum[il + i + j * knsupc],
                                         ready_lsum[maxrecvsz * k * 2 + ii * maxrecvsz + i + j * knsupc]);
                        tmp_sum += ready_lsum[maxrecvsz * lib * 2 + ii * maxrecvsz + i + j * knsupc];
                        //printf("data1-(%d,%d,%d),lib=%d,k=%d,ii=%d,ready_lsum[%d]=%f\n", mype, bid, tid, lib, k, ii,
                        //       maxrecvsz * lib * 2 + ii * maxrecvsz + i + j * knsupc,
                        //       ready_lsum[maxrecvsz * lib * 2 + ii * maxrecvsz + i + j * knsupc]);
                    }

                }
                printf("sum1-(%d,%d,%d),lib=%d,k=%d,sum=%f\n", mype, bid, tid, lib, k, tmp_sum);
                fmod_tmp = atomicSub(&fmod[lib * aln_i], 1);
            }
        }

        volatile int tmp1;
        do{
            tmp1=fmod[lib*aln_i];
            __threadfence();
        }while(tmp1>0);
        printf("fmod done-(%d,%d,%d),lib=%d,k=%d,cnt=%d,sigofset=%d\n",mype,bid,tid,lib,k,cnt,k*2);


        //YL: this thread forwards the lsum subvector using NVSHMEM
        if(LRtree_ptr[lib].myRoot_ != LRtree_ptr[lib].myRank_){
            //cnt=LRtree_ptr[lib].msgSize_;
            my_flag_rd[k*RDMA_FLAG_SIZE]=k;
            my_flag_rd[k*RDMA_FLAG_SIZE+1]=LRtree_ptr[lib].msgSize_;
            RHS_ITERATE(j) {
                for (int i = 0; i < knsupc; i++) {
                    ready_lsum[k * maxrecvsz * 2 + i +j * knsupc] = lsum[il + i+j * knsupc];
                    //printf("data3-(%d,%d,%d),lib=%d,k=%d,i=%d,ready_lsum[%d]=%f\n", mype, bid, tid, lib, k, i,
                    //       k * maxrecvsz * 2 + i +j * knsupc,
                    //       ready_lsum[k * maxrecvsz * 2 + i +j * knsupc]);

                }
            }
            printf("(%d,%d,%d),lib=%d,k=%d,myflagrd=%d,%d\n",mype,bid,tid,lib,k,my_flag_rd[k*RDMA_FLAG_SIZE],my_flag_rd[k*RDMA_FLAG_SIZE+1]);
            C_RdTree_forwardMessageSimple_Device(&LRtree_ptr[lib], (int*)flag_rd_q, &my_flag_rd[RDMA_FLAG_SIZE*k], mype, bid, tid, &ready_lsum[0],maxrecvsz);
        }
    }

} /* dlsum_fmod_inv_gpu_mrhs */
 void dlsum_fmod_inv_gpu_wrap
 (
  int_t nbcol_loc,    /*number of local supernode columns*/
  int_t nbrow_loc,    /*number of local supernode rows*/
  int_t nthread_x,     /*kernel launch parameter*/
  int_t nthread_y,     /*kernel launch parameter*/
  double *lsum,    /* Sum of local modifications.                        */
  double *x,       /* X array (local)                                    */
  int   nrhs,      /* Number of right-hand sides.                        */
  int   maxsup,      /* Max supernode size.                        */
  int_t   nsupers,      /* Number of total supernodes.                        */
  int_t *fmod,     /* Modification count for L-solve.                    */
  C_Tree  *LBtree_ptr,
  C_Tree  *LRtree_ptr,
  int_t *ilsum,
  int_t *Lrowind_bc_dat,   
  long int *Lrowind_bc_offset,      
  double *Lnzval_bc_dat,     
  long int *Lnzval_bc_offset,     
  double *Linv_bc_dat,     
  long int *Linv_bc_offset,     
  int_t *Lindval_loc_bc_dat,     
  long int *Lindval_loc_bc_offset,     
  int_t *xsup,
  gridinfo_t *grid,
  int_t maxrecvsz,
  int* flag_bc_q,
  int* flag_rd_q,
  double* ready_x,
  double* ready_lsum,
  int* my_flag_bc,
  int* my_flag_rd,
  int* d_launch_flag,
  int* d_nfrecv,
  int* h_nfrecv,
  int* d_status,
  int* d_colnum,
  int* d_mynum,
  int* d_mymaskstart,
  int* d_mymasklength,
  int* d_nfrecvmod,
  int* d_statusmod,
  int* d_colnummod,
  int* d_mynummod,
  int* d_mymaskstartmod,
  int* d_mymasklengthmod,
  int* d_recv_cnt,
  int* d_msgnum
 ) {

     gpuStream_t sid = 0;
     int gid = 0;
     int mycol;
     int_t lk, k, knsupc;
     //int_t nblock_ex=CEILING( nbrow_loc, nthread_x*nthread_y);
     int_t nblock_ex = CEILING(nbrow_loc, ((nthread_x * nthread_y) / 32)); //32 (warp) * 8 =256
     //fflush(stdout);
     int mype, npes;
     mype = nvshmem_my_pe();
     npes = nvshmem_n_pes();
     printf("(%d) nbcol_loc %d, nblock_ex %d\n",mype,nbcol_loc,nblock_ex);
     int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
     CUDA_CHECK(hipSetDevice(mype_node));
     int totalth = nthread_x * nthread_y;
     dim3 dimGrid_nv(h_nfrecv[2]);
     dim3 dimBlock_nv(h_nfrecv[1]);
     dim3 dimGrid(nbcol_loc + nblock_ex);
     dim3 dimBlock(nthread_x, nthread_y);

     hipStream_t stream[2];
     for (int i = 0; i < 2; ++i) {
         hipStreamCreate(&stream[i]);
     }

     void *args[] = {&nrhs, &nbcol_loc, &LBtree_ptr, &LRtree_ptr, &maxrecvsz, &mype, &flag_bc_q, &flag_rd_q,
                     &ready_x, &ready_lsum, &my_flag_bc, &my_flag_rd, &totalth, &d_nfrecv, &d_status, &d_launch_flag,
                     &d_colnum, &d_mynum, &d_mymaskstart, &d_mymasklength,
                     &d_nfrecvmod, &d_statusmod, &d_colnummod, &d_mynummod, &d_mymaskstartmod, &d_mymasklengthmod,
                     &d_recv_cnt, &d_msgnum};
     NVSHMEM_CHECK(
             nvshmemx_collective_launch((const void *) schedule, dimGrid_nv, dimBlock_nv, args, 0, stream[0]));
     int launch_success = 0;
     do {
         hipMemcpyAsync(&launch_success, d_launch_flag, 1 * sizeof(int), hipMemcpyDeviceToHost, stream[1]);
     } while (launch_success == 0);
     //int launch_success = 1;
     if (launch_success == 1) {
         dlsum_fmod_inv_gpu_mrhs_nvshmem<<< dimGrid, dimBlock, 0, stream[1] >>>(nbcol_loc, nblock_ex,
                                                                                              lsum, x,
                                                                                              nrhs, maxsup, nsupers,
                                                                                              fmod,
                                                                                              LBtree_ptr, LRtree_ptr,
                                                                                              ilsum,
                                                                                              Lrowind_bc_dat,
                                                                                              Lrowind_bc_offset,
                                                                                              Lnzval_bc_dat,
                                                                                              Lnzval_bc_offset,
                                                                                              Linv_bc_dat,
                                                                                              Linv_bc_offset,
                                                                                              Lindval_loc_bc_dat,
                                                                                              Lindval_loc_bc_offset,
                                                                                              xsup,
                                                                                              grid, maxrecvsz,
                                                                                              mype, flag_bc_q,
                                                                                              flag_rd_q,
                                                                                              ready_x, ready_lsum,
                                                                                              my_flag_bc, my_flag_rd,
                                                                                              totalth, d_launch_flag,
                                                                                              d_nfrecv, d_status,
                                                                                              d_statusmod);


     }
 }
 
#ifdef __cplusplus
}
#endif
