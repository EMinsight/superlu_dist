#include "hip/hip_runtime.h"
/*! \file
Copyright (c) 2003, The Regents of the University of California, through
Lawrence Berkeley National Laboratory (subject to receipt of any required
approvals from U.S. Dept. of Energy)

All rights reserved.

The source code is distributed under BSD license, see the file License.txt
at the top-level directory.
*/


/*! @file
 * \brief Solves a system of distributed linear equations A*X = B with a
 * general N-by-N matrix A using the LU factors computed previously.
 *
 * <pre>
 * -- Distributed SuperLU routine (version 6.1) --
 * Lawrence Berkeley National Lab, Univ. of California Berkeley.
 * October 15, 2008
 * September 18, 2018  version 6.0
 * February 8, 2019  version 6.1.1
 * </pre>
 */

#include <math.h>
#include "superlu_ddefs.h"
#ifndef CACHELINE
#define CACHELINE 64  /* bytes, Xeon Phi KNL, Cori haswell, Edision */
#endif

#ifndef MAXSUPER
#define MAXSUPER 1024  
#endif

#include <stdio.h>
#include "mpi.h"
#include <nvshmem.h>
#include <nvshmemx.h>
#include <stdlib.h>
#include <sched.h>
#include <nvml.h>
#include <omp.h>
#include <hip/hip_cooperative_groups.h>

#undef CUDA_CHECK
#define CUDA_CHECK(stmt)                                                          \
     do {                                                                          \
         hipError_t result = (stmt);                                              \
         if (hipSuccess != result) {                                              \
             fprintf(stderr, "[%s:%d] cuda failed with %s \n", __FILE__, __LINE__, \
                     hipGetErrorString(result));                                  \
             exit(-1);                                                             \
         }                                                                         \
         assert(hipSuccess == result);                                            \
     } while (0)

#undef MPI_CHECK
#define MPI_CHECK(stmt)                                 \
 do {                                                    \
     int result = (stmt);                                \
     if (MPI_SUCCESS != result) {                        \
         fprintf(stderr, "[%s:%d] MPI failed with error %d \n",\
          __FILE__, __LINE__, result);                   \
         exit(-1);                                       \
     }                                                   \
 } while (0)

#define NVSHMEM_CHECK(stmt)                               \
 do {                                                    \
     int result = (stmt);                                \
     if (hipSuccess != result) {                      \
         fprintf(stderr, "[%s:%d] nvshmem failed with error %d \n",\
          __FILE__, __LINE__, result);                   \
         exit(-1);                                       \
     }                                                   \
 } while (0)


 
 

#ifdef __cplusplus
extern "C" {
#endif


// #define USESHARE1RHS 1


/***************************************************************************//**
	 Does sum reduction of n-element array x, leaving total in x[0].
	 Contents of x are destroyed in the process.
	 With k threads, can reduce array up to 2*k in size.
	 Assumes number of threads <= 1024 (which is max number of threads up to CUDA capability 3.0)
	 Having n as template parameter allows compiler to evaluate some conditions at compile time.
	 Calls __syncthreads before & after reduction.
	 @ingroup magma_kernel
 *******************************************************************************/
__device__ void
magma_sum_reduce( int n, int i, double* x )
{
    __syncthreads();
    if ( n > 1024 ) { if ( i < 1024 && i + 1024 < n ) { x[i] += x[i+1024]; }  __syncthreads(); }
    if ( n >  512 ) { if ( i <  512 && i +  512 < n ) { x[i] += x[i+ 512]; }  __syncthreads(); }
    if ( n >  256 ) { if ( i <  256 && i +  256 < n ) { x[i] += x[i+ 256]; }  __syncthreads(); }
    if ( n >  128 ) { if ( i <  128 && i +  128 < n ) { x[i] += x[i+ 128]; }  __syncthreads(); }
    if ( n >   64 ) { if ( i <   64 && i +   64 < n ) { x[i] += x[i+  64]; }  __syncthreads(); }
    if ( n >   32 ) { if ( i <   32 && i +   32 < n ) { x[i] += x[i+  32]; }  __syncthreads(); }
    // probably don't need __syncthreads for < 16 threads
    // because of implicit warp level synchronization.
    if ( n >   16 ) { if ( i <   16 && i +   16 < n ) { x[i] += x[i+  16]; }  __syncthreads(); }
    if ( n >    8 ) { if ( i <    8 && i +    8 < n ) { x[i] += x[i+   8]; }  __syncthreads(); }
    if ( n >    4 ) { if ( i <    4 && i +    4 < n ) { x[i] += x[i+   4]; }  __syncthreads(); }
    if ( n >    2 ) { if ( i <    2 && i +    2 < n ) { x[i] += x[i+   2]; }  __syncthreads(); }
    if ( n >    1 ) { if ( i <    1 && i +    1 < n ) { x[i] += x[i+   1]; }  __syncthreads(); }
}
// end sum_reduce



/******************************************************************************/
static __device__ void
gemv_device_dlsum_fmod(
        int_t m, int_t n, double alpha,
        const double * __restrict__ A, int_t lda,
        const double * __restrict__ x, int_t incx, double beta,
        double       * __restrict__ y, int_t incy)
{
    if (m <= 0 || n <= 0) return;

    int_t num_threads = DIM_X * DIM_Y;
    int_t thread_id = threadIdx_x + threadIdx_y * blockDim_x;

    // threads are all configurated locally
    int_t tx = thread_id % DIM_X;
    int_t ty = thread_id / DIM_X;

    int_t ind = tx;

    __shared__ double sdata[DIM_X * DIM_Y];


    int_t st = 0;

    int_t ed = min(st+m, CEILING(m,DIM_X)*DIM_X);

    int_t iters = CEILING(ed-st,DIM_X) ;

    double zero = 0.0;

    for (int_t i=0; i < iters; i++)
    {
        if (ind < m ) A += ind;

        double res = zero;

        if (ind < m )
        {
            for (int_t col=ty; col < n; col += DIM_Y)
            {
                res += A[col*lda] * x[col*incx];
            }
        }

        if (DIM_X >= num_threads) // indicated 1D threads configuration. Shared memory is not needed, reduction is done naturally
        {
            if (ty == 0 && ind < m)
            {
                y[ind*incy] = alpha*res + beta*y[ind*incy];
            }
        }
        else
        {
            sdata[ty + tx * DIM_Y] = res;

            __syncthreads();

            if ( DIM_Y > 16)
            {
                magma_sum_reduce(DIM_Y, ty, sdata + tx * DIM_Y);
            }
            else
            {
                if (ty == 0 && ind < m)
                {
                    for (int_t i=1; i < DIM_Y; i++)
                    {
                        sdata[tx * DIM_Y] += sdata[i + tx * DIM_Y];
                    }
                }
            }

            if (ty == 0 && ind < m)
            {
                y[ind*incy] = alpha*sdata[tx * DIM_Y] + beta*y[ind*incy];
            }

            __syncthreads();
        }

        if ( ind < m) A -= ind;

        ind += DIM_X;
    }
}





/******************************************************************************/
static __device__
void gemm_device_dlsum_fmod(
        int_t M, int_t N, int_t K,
        int_t blx, int_t bly,
        const double* __restrict__ A, int_t LDA,
        const double* __restrict__ B, int_t LDB,
        double rC[THR_N][THR_M],
        double alpha, double beta)
{
    // #if (__CUDA_ARCH__ >= 200)
    int_t idx = threadIdx_x;  // thread's m dimension
    int_t idy = threadIdx_y;  // thread's n dimension

    int_t idt = DIM_X * idy + idx;    // thread's global number

    int_t idxA = idt % DIM_XA;    // idx within A
    int_t idyA = idt / DIM_XA;    // idy within A

    int_t idxB = idt % DIM_XB;    // idx within B
    int_t idyB = idt / DIM_XB;    // idy within B

    // int_t blx = blockIdx_x;   // block's m dimension
    // int_t bly = blockIdx_y;   // block's n dimension

    __shared__ double sA[BLK_K][BLK_M+1];      // +1 only required if A is transposed
    __shared__ double sB[BLK_N][BLK_K+1];      // +1 always required

    // Registers for the innermost loop
    double rA[THR_M];
    double rB[THR_N];

    double ra[BLK_K/DIM_YA+1][BLK_M/DIM_XA];
    double rb[BLK_N/DIM_YB][BLK_K/DIM_XB+1];

    const double *offs_dA = A + blx*BLK_M     + idyA*LDA + idxA;
    const double *offs_dB = B + bly*BLK_N*LDB + idyB*LDB + idxB;
    int_t boundA = (LDA*(K-1) + M) - ( blx*BLK_M  + idyA*LDA + idxA ) -1;
    int_t boundB = (LDB*(N-1) + K) - ( bly*BLK_N*LDB + idyB*LDB + idxB ) -1;

    int_t m, n, k, kk;
    double zero = 0.0;

    // Zero C
#pragma unroll
    for (n = 0; n < THR_N; n++)
#pragma unroll
            for (m = 0; m < THR_M; m++)
                rC[n][m] = zero;

#pragma unroll
    for (n = 0; n < BLK_K; n += DIM_YA)
#pragma unroll
            for (m = 0; m < BLK_M; m += DIM_XA)
                sA[n+idyA][m+idxA] = fetch(A, m, n, boundA);

#pragma unroll
    for (n = 0; n < BLK_N; n += DIM_YB)
#pragma unroll
            for (m = 0; m < BLK_K; m += DIM_XB)
                sB[n+idyB][m+idxB] = fetch(B, m, n, boundB);

    __syncthreads();

    for (kk = 0; kk < K-BLK_K; kk += BLK_K)
    {
        offs_dA += BLK_K*LDA;
        boundA  -= BLK_K*LDA;

        offs_dB += BLK_K;
        boundB  -= BLK_K;

#pragma unroll
        for (n = 0; n < BLK_K/DIM_YA; n++)
#pragma unroll
                for (m = 0; m < BLK_M/DIM_XA; m++)
                    ra[n][m] = fetch(A, m*DIM_XA, n*DIM_YA, boundA);

#pragma unroll
        for (n = 0; n < BLK_N/DIM_YB; n++)
#pragma unroll
                for (m = 0; m < BLK_K/DIM_XB; m++)
                    rb[n][m] = fetch(B, m*DIM_XB, n*DIM_YB, boundB);

        // Multiply
#pragma unroll
        for (k = 0; k < BLK_K; k++)
        {
            // Load A shmem->regs
#pragma unroll
            for (m = 0; m < THR_M; m++)
                rA[m] = sA[k][m*DIM_X+idx];

            // Load B shmem->regs
#pragma unroll
            for (n = 0; n < THR_N; n++)
                rB[n] = sB[n*DIM_Y+idy][k];

            // Compute
#pragma unroll
            for (n = 0; n < THR_N; n++) {
#pragma unroll
                for (m = 0; m < THR_M; m++) {
                    fma(rA[m], rB[n], rC[n][m]);
                }
            }
        }

        __syncthreads();

#pragma unroll
        for (n = 0; n < BLK_K/DIM_YA; n++)
#pragma unroll
                for (m = 0; m < BLK_M/DIM_XA; m++)
                    sA[n*DIM_YA+idyA][m*DIM_XA+idxA] = ra[n][m];

#pragma unroll
        for (n = 0; n < BLK_N/DIM_YB; n++)
#pragma unroll
                for (m = 0; m < BLK_K/DIM_XB; m++)
                    sB[n*DIM_YB+idyB][m*DIM_XB+idxB] = rb[n][m];

        __syncthreads();
    }

    // Multiply last full (BLK_K) or partial block of
    // columns of op(A) and rows of op(B).
    // It's okay that m,n exceed matrix bounds as all work is in registers
    // or shared memory, and out-of-bounds rC[n][m] will not be saved later.
    kk = K - kk;
#pragma unroll
    for (k = 0; k < kk; k++)
    {
        // Load A shmem->regs
#pragma unroll
        for (m = 0; m < THR_M; m++)
            rA[m] = sA[k][m*DIM_X+idx];

        // Load B shmem->regs
#pragma unroll
        for (n = 0; n < THR_N; n++)
            rB[n] = sB[n*DIM_Y+idy][k];

        // Compute
#pragma unroll
        for (n = 0; n < THR_N; n++) {
#pragma unroll
            for (m = 0; m < THR_M; m++) {
                fma(rA[m], rB[n], rC[n][m]);
            }
        }
    }

    // Store C regs->dev
    // if( beta == make_FloatingPoint_t(0.0,0.0) ) {
    // #pragma unroll
    // for (n = 0; n < THR_N; n++) {
    // int_t coord_dCn = bly*BLK_N + n*DIM_Y + idy;
    // #pragma unroll
    // for (m = 0; m < THR_M; m++) {
    // int_t coord_dCm = blx*BLK_M + m*DIM_X + idx;
    // if (coord_dCm < M && coord_dCn < N) {
    // int_t offsC = coord_dCn*LDC + coord_dCm;

    // double &regC = rC[n][m];
    // double &memC = C[offsC];

    // // memC = mul(alpha, regC);
    // }
    // }
    // }
    // } else {
    // #pragma unroll
    // for (n = 0; n < THR_N; n++) {
    // int_t coord_dCn = bly*BLK_N + n*DIM_Y + idy;
    // #pragma unroll
    // for (m = 0; m < THR_M; m++) {
    // int_t coord_dCm = blx*BLK_M + m*DIM_X + idx;
    // if (coord_dCm < M && coord_dCn < N) {
    // int_t offsC = coord_dCn*LDC + coord_dCm;

    // double &regC = rC[n][m];
    // double &memC = C[offsC];

    // // memC = add(mul(alpha, regC), mul(beta, memC));
    // }
    // }
    // }
    // }
    // #endif /* (__CUDA_ARCH__ >= 200) */
}
#define cudaCheckError() { \
    hipError_t e=hipGetLastError();                           \
    if(e!=hipSuccess) {                       \
        printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));                           \
        exit(EXIT_FAILURE);                   \
    }                       \
}

void checkP2Paccess(int numGPUs)
{
    for (int i = 0; i < numGPUs; i++) {
        hipSetDevice(i);
        cudaCheckError();

        for (int j = 0; j < numGPUs; j++) {
            int access;
            if (i != j) {
                hipDeviceCanAccessPeer(&access, i, j);
                cudaCheckError();
                printf("Device=%d %s Access Peer Device=%d\n", i, access ? "CAN" : "CANNOT", j);
            }
        }
    }
    printf("\n***NOTE: In case a device doesn't have P2P access to other one, it falls back to normal memcopy procedure.\nSo you can see lesser Bandwidth (GB/s) and unstable Latency (us) in those cases.\n\n");
}


void nv_init_wrapper(int* c, char *v[], int* omp_mpi_level)
{
    int *target;
    int rank, nranks, ndevices;
    MPI_Comm mpi_comm;
    nvshmemx_init_attr_t attr;
    int mype, npes, mype_node;
    //MPI_CHECK(MPI_Init(&c, &v));
    MPI_CHECK(MPI_Init_thread( c, &v, MPI_THREAD_MULTIPLE, omp_mpi_level));
    MPI_CHECK(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    MPI_CHECK(MPI_Comm_size(MPI_COMM_WORLD, &nranks));


    //CUDA_CHECK(hipSetDevice(rank%ndevices));
    mpi_comm = MPI_COMM_WORLD;
    attr.mpi_comm = &mpi_comm;
    NVSHMEM_CHECK(nvshmemx_init_attr (NVSHMEMX_INIT_WITH_MPI_COMM, &attr));
    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();

    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    char name[MPI_MAX_PROCESSOR_NAME];
    int resultlength;
    MPI_CHECK(MPI_Get_processor_name(name, &resultlength));
    int get_cur_dev;
    CUDA_CHECK(hipGetDeviceCount(&ndevices));
    //CUDA_CHECK(hipSetDevice(rank%ndevices));
    CUDA_CHECK(hipSetDevice(mype_node));
    CUDA_CHECK(hipGetDevice(&get_cur_dev));

    hipDeviceProp_t prop;
    //CUDA_CHECK(hipGetDeviceProperties(&prop, rank%ndevices));
    CUDA_CHECK(hipGetDeviceProperties(&prop, mype_node));
    printf("** MPI %d/%d, NVSHMEM %d/%d, mype_node=%d, device name: %s bus id: %d, "
           "ndevices=%d,cur=%d, node=%s **\n",
           rank,nranks,mype,npes,mype_node, prop.name, prop.pciBusID,
           ndevices,get_cur_dev,name);
    fflush(stdout);

}


__device__ void C_BcTree_forwardMessageSimple_Device(C_Tree* tree,  int* flag_bc_q,  int* my_flag_bc, int mype, int tid,double* ready_x){
    //int BCsendoffset;
    int sig = 1;
    for( int idxRecv = 0; idxRecv < tree->destCnt_; ++idxRecv ) {
        int iProc = tree->myDests_[idxRecv];
        //BCsendoffset = my_flag_bc[2];
        //double sum=0;
        //if (tid==0) {
        //    for(int i=0;i<my_flag_bc[3];i++){
        //        //printf("(%d), data, %d,%lf\n",mype,i,ready_x[i]);
        //        sum+=ready_x[my_flag_bc[2]+i];
        //    }
        //    printf("Start (%d), forwardDevice, send to %d, signal offset=%d, msgsz=%d,sum=%lf\n",mype,iProc,my_flag_bc[0],my_flag_bc[3],sum);
        //}
        //__syncthreads();
        //if(tid==0) printf("Start (%d), forwardDevice, send to %d, signal offset=%d, data offset=%d, msgsz=%d\n",mype,iProc,my_flag_bc[1],BCsendoffset,my_flag_bc[3]);
        //__syncthreads();
        //nvshmemx_double_put_block(&ready_x[BCsendoffset],ready_x,my_flag_bc[3],iProc);
        nvshmemx_double_put_nbi_block(&ready_x[my_flag_bc[2]], &ready_x[my_flag_bc[2]], my_flag_bc[3], iProc);
        //nvshmem_double_put_nbi(ready_x, &ready_x[0], my_flag_bc[3], iProc);
        //nvshmem_double_put(&ready_x[BCsendoffset],ready_x,my_flag_bc[3],iProc);
        //nvshmem_quiet();
        nvshmem_fence();
        //__syncthreads();
        if (tid == 0) {
            nvshmemx_int_signal((int*)(flag_bc_q + my_flag_bc[0]), sig, iProc);
            //nvshmem_quiet();
            //printf("Done (%d), forwardDevice, send to %d, signal offset=%d, data offset=%d, msgsz=%d\n", mype, iProc,
            //       my_flag_bc[0], my_flag_bc[2], my_flag_bc[3]);

        }
    }
}

__device__ void C_RdTree_forwardMessageSimple_Device(C_Tree* Tree, int* flag_rd_q, int* my_flag_rd, int mype, int bid, int tid, double* ready_lsum, int maxrecvsz){
    int data_ofset,sig_ofset;
    if(Tree->myIdx %2 ==0){
        sig_ofset = my_flag_rd[0]*2;
        data_ofset = my_flag_rd[0]*maxrecvsz*2;
    }else{
        sig_ofset = my_flag_rd[0]*2+1;
        data_ofset = my_flag_rd[0]*maxrecvsz*2+maxrecvsz;
    }
    if(Tree->myRank_!=Tree->myRoot_){
        ////forward to my root if I have received everything
        int iProc = Tree->myRoot_;
        //// YL: Use NVSHMEM to send to my parent
        //printf("forwardMessage-(%d,%d,%d), rd--1, send to %d,data_ofset=%d,maxrecvsz=%d,k=%d,sigoffset=%d\n",mype,bid, tid,iProc,data_ofset,maxrecvsz,my_flag_rd[0],sig_ofset);
        //__device__ void nvshmem_TYPENAME_put(TYPE *dest, const TYPE *source, size_t nelems, int pe)
        //double sum=0;
        //for(int i=my_flag_rd[0]*maxrecvsz*2;i<my_flag_rd[0]*maxrecvsz*2+my_flag_rd[1];i++){
        //    //printf("(%d), data, %d\n",mype,i);
        //    //printf("(%d), data, %d,%lf\n",mype,i,ready_lsum[i]);
        //    sum+=ready_lsum[i];
        //}
        //printf("forwardMessage- (%d), forwardDevice, send to %d, lib=%d,sum=%lf\n",mype,iProc,my_flag_rd[0],sum);

        nvshmem_double_put_nbi(&ready_lsum[data_ofset],&ready_lsum[my_flag_rd[0]*maxrecvsz*2],my_flag_rd[1],iProc);
        nvshmem_fence();
        int sig=1;
        nvshmemx_int_signal((int*)flag_rd_q+sig_ofset, sig, iProc);
        //printf("forwardMessage-(%d,%d,%d), rd--2, send to %d,data_ofset=%d,sig_ofset=%d\n",mype,bid, tid,iProc,data_ofset,sig_ofset);
    }
}


__global__ void ping_pong(volatile double *data_d, volatile int *flag_d, volatile int *flag_d_local,
                          int len, int pe, int iter, int skip, int peer) {
    long long int start, stop;
    double usec, time,bw;
    int i, tid,bid;

    tid = threadIdx.x;
    bid = blockIdx.x;

    //nvshmem_barrier_all();
    //if (tid == 0) printf("(%d) iter=%d, skip=%d,peer=%d\n", pe, iter, skip, peer);
    for (i = 0; i < (iter + skip); i++) {
        if (i == skip) start = clock64();
        if (pe == peer) {
            if (tid==0) printf("(%d,%d) mypeer=%d\n",pe,bid,peer);
            if (tid == 0) nvshmem_int_wait_until((int *) flag_d + bid, NVSHMEM_CMP_EQ, bid + (i + 1));
            if (tid==0) printf("(%d,%d) mypeer=%d,msg arrived\n",pe,bid,peer);

            if (tid==0) nvshmem_double_put_nbi((double *)data_d+bid, (double *)data_d+bid, len, peer);
            //nvshmemx_double_put_nbi_block((double *) data_d + bid, (double *) data_d + bid, len, 0);
            //__syncthreads();
            if (tid==0) printf("(%d,%d) mypeer=%d,send data done\n",pe,bid,peer);

            if (tid==0) nvshmem_fence();
            if (tid == 0) nvshmemx_int_signal((int *) flag_d + bid, bid + (i + 1), 0);
            if (tid==0) printf("(%d,%d) mypeer=%d, signal done\n",pe,bid,peer);
        } else if (pe == 0) {
            //nvshmem_double_put_nbi((double *) data_d + bid, (double *) data_d + bid, len, peer);
            if (tid==0) nvshmemx_double_put_nbi_block((double *) data_d + bid, (double *) data_d + bid, len, peer);
            if (tid==0) printf("(%d,%d) mypeer=%d,send data done\n",pe,bid,peer);

            if (tid==0) nvshmem_fence();
            if (tid == 0) {
                nvshmemx_int_signal((int *) flag_d + bid, bid + (i + 1), peer);
                printf("(%d,%d) mypeer=%d, signal done\n",pe,bid,peer);
                do{
                    printf("(%d,%d) checking flag=%d\n",pe,bid,flag_d[bid]);
                }while(flag_d[bid]!=1);
                nvshmem_int_wait_until((int *) flag_d + bid, NVSHMEM_CMP_EQ, bid + (i + 1));
                if (tid==0) printf("(%d,%d) mypeer=%d,msg arrived\n",pe,bid,peer);
            }
        }
    }
    //nvshmem_barrier_all();
    stop = clock64();
    nvshmem_quiet();

    if ((pe == 0) && !tid) {
        time = (stop - start) / iter;
        usec = time * 1000 / clockrate;
        bw = ((float) iter * (float) len * sizeof(double) * 2 * clockrate) / ((time / 1000) * 1024 * 1024 * 1024);
        printf("Block (%d) %7lu, %8.2f, %4.6f \n", bid, len * sizeof(int), usec, bw);
    }
}

__global__ void schedule
(
int nrhs,
C_Tree  *LRtree_ptr,
int_t maxrecvsz,
int mype,
int* flag_bc_q,
int* flag_rd_q,
double* ready_x,
double* ready_lsum,
int* my_flag_bc,
int* my_flag_rd,
int* d_nfrecv,
int* d_status,
int* d_launch_flag,
int* d_colnum,
int* d_mynum,
int* d_mymaskstart,
int* d_mymasklength,
int* d_nfrecvmod,
int* d_statusmod,
int* d_colnummod,
int* d_mynummod,
int* d_mymaskstartmod,
int* d_mymasklengthmod,
int* d_recv_cnt,
int* d_msgnum,
double *lsum,    /* Sum of local modifications.                        */
int_t *fmod,     /* Modification count for L-solve.                    */
gridinfo_t *grid,
int_t *xsup,
int_t *ilsum,
int nbrow_loc,
int_t  nsupers
) {

    int bid = blockIdx.x;
    //int global_id= blockIdx.x * blockDim.x * blockDim.y + threadIdx.x + threadIdx.y * blockDim.x;
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    d_launch_flag[0] = 1;
    int WAIT_NUM_THREADS = d_nfrecv[1]*d_nfrecv[2];
    //if (tid==0) printf("(%d) WAIT_NUM_THREADS=%d,tot_wait_col=%d\n",mype,WAIT_NUM_THREADS,d_nfrecv[0]);

   if (bid == 0) { // for BC
        if (WAIT_NUM_THREADS >= d_nfrecv[0]) {
            if (tid < d_nfrecv[0]) {
                //printf("WAIT1 (%d,%d) wait for col %d,flag=%d\n", mype, tid, d_colnum[tid],flag_bc_q[d_colnum[tid]]);
                nvshmem_int_wait_until((int *) flag_bc_q + d_colnum[tid], NVSHMEM_CMP_EQ, 1);
                d_status[d_colnum[tid]] = 1;
                //printf("WAIT1 (%d,%d) msg arrived in col %d\n", mype, tid, d_colnum[tid]);
            }
        } else {
            int delta = d_nfrecv[0] % WAIT_NUM_THREADS;
            if (tid < delta) {
                d_mynum[tid] = d_nfrecv[0] / WAIT_NUM_THREADS + 1;
            } else {
                d_mynum[tid] = d_nfrecv[0] / WAIT_NUM_THREADS;
            }
            __syncthreads();
            d_mymaskstart[tid] = 0;
            for (int i = 0; i < tid; i++) {
                d_mymaskstart[tid] += d_mynum[i];
            }
            d_mymasklength[tid] = d_colnum[d_mymaskstart[tid] + d_mynum[tid] - 1] - d_colnum[d_mymaskstart[tid]] + 1;
            __syncthreads();
            //printf("WAIT2 (%d,%d) mynum=%d, start=%d,%d length=%d\n",mype,tid,d_mynum[tid],d_mymaskstart[tid],d_colnum[d_mymaskstart[tid]],d_mymasklength[tid]);

            for (int i = 0; i < d_mynum[tid]; i++) {
                int wm_val = nvshmem_int_wait_until_any(flag_bc_q + d_colnum[d_mymaskstart[tid]], d_mymasklength[tid],
                                                        d_status + d_colnum[d_mymaskstart[tid]], NVSHMEM_CMP_EQ, 1);
                d_status[d_colnum[d_mymaskstart[tid]] + wm_val] = 1;
                //printf("WAIT2 (%d,%d) msg arrived in col %d, i=%d\n",mype,tid,d_colnum[d_mymaskstart[tid]] + wm_val, i);
            }
        }
   }
#if 0
    if (bid == 1) { // for RD
        //if (tid==0) printf("RD---(%d) WAIT_NUM_THREADS=%d,tot_wait_col=%d\n",mype,WAIT_NUM_THREADS,d_nfrecvmod[1]);
        int j, iam, lib, mycol, myrow, k, knsupc, il, cnt;
        int_t fmod_tmp, aln_i;
        aln_i = 1;
        double temp;
        if (WAIT_NUM_THREADS >= d_nfrecvmod[1]) { // one thread wait for one col
            if (tid < d_nfrecvmod[1]) {
                //printf("(%d,%d,%d) d_colnummod=%d,recv_cnt=%d\n", mype, bid, tid, d_colnummod[tid], d_recv_cnt[d_colnummod[tid]]);
                for (int i = 0; i < d_recv_cnt[d_colnummod[tid]]; i++) {
                    //printf("(%d,%d,%d) d_colnummod=%d,recv_cnt=%d,i=%d,wait_off=%d,%d,status=%d,%d\n", mype, bid, tid, d_colnummod[tid], d_recv_cnt[d_colnummod[tid]],i,d_colnummod[tid]*2, d_colnummod[tid]*2+1,d_statusmod[d_colnummod[tid]*2], d_statusmod[d_colnummod[tid]*2+1]);
                    int wm_val = nvshmem_int_wait_until_any(flag_rd_q + d_colnummod[tid] * 2, 2,
                                                            d_statusmod + d_colnummod[tid] * 2, NVSHMEM_CMP_EQ, 1);
                    d_statusmod[d_colnummod[tid] * 2 + wm_val] = 1;

                    lib = (d_colnummod[tid] * 2 + wm_val) / 2;

                    iam = grid->iam;
                    mycol = MYCOL(iam, grid);
                    myrow = MYROW(iam, grid);

                    k = myrow + lib * grid->nprow; // global block row
                    knsupc = SuperSize(k);
                    il = LSUM_BLK(lib);
                    cnt = LRtree_ptr[lib].destCnt_;
                    //printf("(%d,%d,%d),idx=%d,lib=%d,cnt=%d\n", mype, bid, tid,
                    //       d_colnummod[tid] * 2 + wm_val, lib, cnt);
                    if (d_statusmod[lib * 2] + d_statusmod[lib * 2 + 1] == cnt) {
                        double tmp_sum = 0;
                        int ii = 0;
                        if (cnt == 2) {
                            for (ii = 0; ii < cnt; ++ii) {
                                tmp_sum = 0;
                                RHS_ITERATE(j) {
                                    for (int aab = 0; aab < knsupc; ++aab) {
                                        //temp=atomicAdd(&lsum[il+i + j*knsupc], ready_lsum[maxrecvsz*lib*2+ii*maxrecvsz + i + j*knsupc]  );
                                        temp = atomicAdd(&lsum[il + aab + j * knsupc],
                                                         ready_lsum[maxrecvsz * lib * 2 + ii * maxrecvsz + aab +
                                                                    j * knsupc]);
                                        tmp_sum += ready_lsum[maxrecvsz * lib * 2 + ii * maxrecvsz + aab + j * knsupc];
                                        //printf("data2-(%d,%d,%d),lib=%d,k=%d,ii=%d,ready_lsum[%d]=%f\n", mype, bid, tid,
                                        //       lib, k, ii,
                                        //       maxrecvsz * lib * 2 + ii * maxrecvsz + i + j * knsupc,
                                        //       ready_lsum[maxrecvsz * lib * 2 + ii * maxrecvsz + i + j * knsupc]);
                                    }

                                    // atomic return old val
                                    fmod_tmp = atomicSub(&fmod[lib * aln_i], 1);
                                    //printf("sum2-(%d,%d,%d),lib=%d,k=%d,sum=%f,fmod_tmp=%d\n", mype, bid, tid, lib, k,
                                    //       tmp_sum,fmod_tmp);
                                }
                            }
                        }
                        if (cnt == 1) {
                            if (flag_rd_q[lib * 2 + 1] == 1) ii = 1;
                            RHS_ITERATE(j) {
                                for (int aab = 0; aab < knsupc; ++aab) {
                                    //temp=atomicAdd(&lsum[il+i + j*knsupc], ready_lsum[maxrecvsz*lib*2+ii*maxrecvsz + i + j*knsupc]  );
                                    temp = atomicAdd(&lsum[il + aab + j * knsupc],
                                                     ready_lsum[maxrecvsz * lib * 2 + ii * maxrecvsz + aab + j * knsupc]);
                                    tmp_sum += ready_lsum[maxrecvsz * lib * 2 + ii * maxrecvsz + aab + j * knsupc];
                                    //printf("data1-(%d,%d,%d),lib=%d,k=%d,ii=%d,ready_lsum[%d]=%f\n", mype, bid, tid, lib, k, ii,
                                    //       maxrecvsz * lib * 2 + ii * maxrecvsz + i + j * knsupc,
                                    //       ready_lsum[maxrecvsz * lib * 2 + ii * maxrecvsz + i + j * knsupc]);
                                }

                            }
                            // atomic return old val
                            fmod_tmp = atomicSub(&fmod[lib * aln_i], 1);
                            //printf("sum1-(%d,%d,%d),lib=%d,k=%d,sum=%f,fmod_tmp=%d\n", mype, bid, tid, lib, k, tmp_sum,fmod_tmp);
                        }

                        if (fmod_tmp == 1) {// forward RD
                            //senddone[lk]=1;
                            if (LRtree_ptr[lib].myRoot_ != LRtree_ptr[lib].myRank_) {
                                //cnt=LRtree_ptr[lib].msgSize_;
                                my_flag_rd[k * RDMA_FLAG_SIZE] = lib;
                                my_flag_rd[k * RDMA_FLAG_SIZE + 1] = LRtree_ptr[lib].msgSize_;
                                RHS_ITERATE(j) {
                                    for (int aab = 0; aab < knsupc; aab++) {
                                        ready_lsum[lib * maxrecvsz * 2 + aab + j * knsupc] = lsum[il + aab + j * knsupc];
                                        //printf("data3-(%d,%d,%d),lib=%d,k=%d,i=%d,ready_lsum[%d]=%f\n", mype, bid, tid, lib, k, i,
                                        //       k * maxrecvsz * 2 + i +j * knsupc,
                                        //       ready_lsum[k * maxrecvsz * 2 + i +j * knsupc]);

                                    }
                                }
                                //printf("(%d,%d,%d),in wait lib=%d,k=%d,myflagrd=%d,%d\n", mype, bid, tid, lib, k,
                                //       my_flag_rd[k * RDMA_FLAG_SIZE], my_flag_rd[k * RDMA_FLAG_SIZE + 1]);
                                C_RdTree_forwardMessageSimple_Device(&LRtree_ptr[lib], (int *) flag_rd_q,
                                                                     &my_flag_rd[RDMA_FLAG_SIZE * k], mype, bid, tid,
                                                                     &ready_lsum[0], maxrecvsz);
                            }
                        }
                    }
                }//for
            }
        } else {
            int delta = d_nfrecvmod[1] % WAIT_NUM_THREADS;

            if (tid < delta) {
                d_mynummod[tid] = d_nfrecvmod[1] / WAIT_NUM_THREADS+1;
            } else {
                d_mynummod[tid] = d_nfrecvmod[1] / WAIT_NUM_THREADS;
            }
            __syncthreads();

            d_mymaskstartmod[tid] = 0;
            d_mymasklengthmod[tid] = 0;
            d_msgnum[tid] = 0;

            //d_mymaskstartmod: start offset of d_colnummod
            for (int i = 0; i < tid; i++) {
                d_mymaskstartmod[tid] += d_mynummod[i];
                //printf("(%d,%d,%d),i=%d,d_mynummod=%d,d_mymaskstartmod=%d\n",
                //       mype,bid,tid,i,
                //       d_mynummod[i],d_mymaskstartmod[tid]);
            }
            __syncthreads();

            for (int i = d_mymaskstartmod[tid]; i < d_mymaskstartmod[tid] + d_mynummod[tid]; i++) {
                d_msgnum[tid] += d_recv_cnt[d_colnummod[i]];
                //printf("(%d,%d,%d),i=%d,d_recv_cnt=%d\n",mype,bid,tid,i,d_recv_cnt[d_colnummod[i]]);
            }
            d_mymasklengthmod[tid] = (d_colnummod[d_mymaskstartmod[tid] + d_mynummod[tid] - 1]
                                      - d_colnummod[d_mymaskstartmod[tid]]+1)*2;
            //printf("(%d,%d,%d) waitcol=%d,msgnum=%d,masklength=%d,start=%d\n",mype,bid,tid,
            //                   d_mynummod[tid],d_msgnum[tid],
            //                   d_mymasklengthmod[tid],d_mymaskstartmod[tid]);

            for (int i = 0; i < d_msgnum[tid]; i++) {
                //printf("(%d,%d,%d)--before wait any,i=%d/%d\n",mype,bid,tid,i,d_msgnum[tid]);
                int wm_val = nvshmem_int_wait_until_any(&flag_rd_q[d_colnummod[d_mymaskstartmod[tid]] * 2],
                                                        d_mymasklengthmod[tid],
                                                        &d_statusmod[d_colnummod[d_mymaskstartmod[tid]] * 2],
                                                        NVSHMEM_CMP_EQ, 1);
                d_statusmod[d_colnummod[d_mymaskstartmod[tid]]*2 + wm_val] = 1;
                lib = (d_colnummod[d_mymaskstartmod[tid]]*2 + wm_val) / 2;
                //printf("(%d,%d,%d)--recv a msg, offset=%d,base=%d, lib=%d, flag=%d,status=%d\n",mype,bid,tid,wm_val,d_colnummod[d_mymaskstartmod[tid]]*2,
                //                  lib,flag_rd_q[d_colnummod[d_mymaskstartmod[tid]] * 2+wm_val],d_statusmod[d_colnummod[d_mymaskstartmod[tid]]*2 + wm_val]);
                iam = grid->iam;
                mycol = MYCOL(iam, grid);
                myrow = MYROW(iam, grid);

                k = myrow + lib * grid->nprow; // global block row
                knsupc = SuperSize(k);
                il = LSUM_BLK(lib);
                cnt = LRtree_ptr[lib].destCnt_;
                //printf("HERE2-(%d,%d,%d),lib=%d,k=%d,wm_val=%d,cnt=%d,%d, mycnt=%d\n", mype, bid, tid, lib, k,
                //       wm_val,cnt,d_recv_cnt[lib],d_statusmod[lib * 2] + d_statusmod[lib * 2 + 1]);

                if (d_statusmod[lib * 2] + d_statusmod[lib * 2 + 1] == cnt) {
                    double tmp_sum = 0;
                    int ii = 0;
                    if (cnt == 2) {
                        for (ii = 0; ii < cnt; ++ii) {
                            tmp_sum = 0;
                            RHS_ITERATE(j) {
                                for (int aab = 0; aab < knsupc; aab++) {
                                    //temp=atomicAdd(&lsum[il+i + j*knsupc], ready_lsum[maxrecvsz*lib*2+ii*maxrecvsz + i + j*knsupc]  );
                                    temp = atomicAdd(&lsum[il + aab + j * knsupc],
                                                     ready_lsum[maxrecvsz * lib * 2 + ii * maxrecvsz + aab +
                                                                j * knsupc]);
                                    tmp_sum += ready_lsum[maxrecvsz * lib * 2 + ii * maxrecvsz + aab + j * knsupc];
                                    //printf("data2-(%d,%d,%d),lib=%d,k=%d,ii=%d,ready_lsum[%d]=%f\n", mype, bid, tid,
                                    //       lib, k, ii,
                                    //       maxrecvsz * lib * 2 + ii * maxrecvsz + i + j * knsupc,
                                    //       ready_lsum[maxrecvsz * lib * 2 + ii * maxrecvsz + i + j * knsupc]);
                                }

                                // atomic return old val
                                fmod_tmp = atomicSub(&fmod[lib * aln_i], 1);
                                //printf("sum2-(%d,%d,%d),lib=%d,k=%d,sum=%f,fmod_tmp=%d\n", mype, bid, tid, lib, k,tmp_sum,fmod_tmp);
                            }
                        }
                    }
                    if (cnt == 1) {
                        if (flag_rd_q[k * 2 + 1] == 1) ii = 1;
                        RHS_ITERATE(j) {
                            for (int aab = 0; aab < knsupc; ++aab) {
                                temp = atomicAdd(&lsum[il + aab + j * knsupc],
                                                 ready_lsum[maxrecvsz * lib * 2 + ii * maxrecvsz + aab + j * knsupc]);
                                tmp_sum += ready_lsum[maxrecvsz * lib * 2 + ii * maxrecvsz + aab + j * knsupc];
                                //printf("data1-(%d,%d,%d),lib=%d,k=%d,ii=%d,ready_lsum[%d]=%f\n", mype, bid, tid, lib, k, ii,
                                //       maxrecvsz * lib * 2 + ii * maxrecvsz + i + j * knsupc,
                                //       ready_lsum[maxrecvsz * lib * 2 + ii * maxrecvsz + i + j * knsupc]);
                            }

                        }
                        // atomic return old val
                        fmod_tmp = atomicSub(&fmod[lib * aln_i], 1);
                        //printf("sum1-(%d,%d,%d),lib=%d,k=%d,sum=%f,fmod_tmp=%d\n", mype, bid, tid, lib, k, tmp_sum,fmod_tmp);
                    }

                    if (fmod_tmp == 1) {// forward RD
                        //printf("sum1-(%d,%d,%d),lib=%d, myRoot=%d\n", mype, bid, tid, lib,LRtree_ptr[lib].myRoot_);
                        if (LRtree_ptr[lib].myRoot_ != LRtree_ptr[lib].myRank_) {
                            //cnt=LRtree_ptr[lib].msgSize_;
                            my_flag_rd[k * RDMA_FLAG_SIZE] = lib;
                            my_flag_rd[k * RDMA_FLAG_SIZE + 1] = LRtree_ptr[lib].msgSize_;
                            RHS_ITERATE(j) {
                                for (int aab = 0; aab < knsupc; aab++) {
                                    ready_lsum[lib * maxrecvsz * 2 + aab + j * knsupc] = lsum[il + aab + j * knsupc];
                                    //printf("data3-(%d,%d,%d),lib=%d,k=%d,i=%d,ready_lsum[%d]=%f\n", mype, bid, tid, lib, k, i,
                                    //       k * maxrecvsz * 2 + i +j * knsupc,
                                    //       ready_lsum[k * maxrecvsz * 2 + i +j * knsupc]);

                                }
                            }
                            //printf("(%d,%d,%d),in wait lib=%d,k=%d,myflagrd=%d,%d\n", mype, bid, tid, lib, k,
                            //       my_flag_rd[k * RDMA_FLAG_SIZE], my_flag_rd[k * RDMA_FLAG_SIZE + 1]);
                            C_RdTree_forwardMessageSimple_Device(&LRtree_ptr[lib], (int *) flag_rd_q,
                                                                 &my_flag_rd[RDMA_FLAG_SIZE * k], mype, bid, tid,
                                                                 &ready_lsum[0], maxrecvsz);
                        }
                    }
                }
            }//for
        } // else WAIT_NUM_THREAD<recv
    }
#endif
}



// /************************************************************************/
// /*! \brief
// *
// * <pre>
// * Purpose
// * =======
// *   Perform local block modifications: lsum[i] -= L_i,k * X[k].
// * </pre>
// */
// __global__ void dlsum_fmod_inv_gpu_1rhs
// /************************************************************************/
// (
// double *lsum,    /* Sum of local modifications.                        */
// double *x,       /* X array (local)                                    */
// double *rtemp,   /* Result of full matrix-vector multiply.             */
// int   nrhs,      /* Number of right-hand sides.                        */
// int   maxsup,      /* Max supernode size.                        */
// int_t   nsupers,      /* Number of total supernodes.                        */
// int_t *fmod,     /* Modification count for L-solve.                    */
// int_t *xsup,
// gridinfo_t *grid,
// LocalLU_t *Llu
// )
// {
// double alpha = 1.0, beta = 0.0,malpha=-1.0;
// double *lusup, *lusup1;
// double *dest;
// double *Linv;/* Inverse of diagonal block */
// int    iam, iknsupc, myrow, mycol, krow, nbrow, nbrow1, nbrow_ref, nsupr, nsupr1, p, pi, idx_r,m;
// int_t  k,i, l,ii,jj, ik, il, ikcol, irow, j, lb, lk, rel, lib,lready;
// int_t  *lsub, *lsub1, nlb1, lptr1, luptr1,*lloc;
// int_t  luptr_tmp,luptr_tmp1,lptr1_tmp,maxrecvsz, idx_i, idx_v,idx_n,  idx_l, fmod_tmp, lbstart,lbend,nn,Nchunk,nlb_loc,remainder;
// int thread_id1;
// flops_t ops_loc=0.0;
// MPI_Status status;
// int test_flag;
// yes_no_t done;
// C_Tree  *LBtree_ptr = Llu->LBtree_ptr;
// C_Tree  *LRtree_ptr = Llu->LRtree_ptr;
// int_t* idx_lsum,idx_lsum1;
// const int Nbk=1;
// // __shared__ double rtemp_loc[128];
// double temp,temp1;
// int_t ldalsum;
// int_t nleaf_send_tmp;
// int_t lptr;      /* Starting position in lsub[*].                      */
// int_t luptr;     /* Starting position in lusup[*].                     */
// int_t iword = sizeof(int_t);
// int_t dword = sizeof (double);
// int_t aln_d,aln_i;
// aln_d = 1;//ceil(CACHELINE/(double)dword);
// aln_i = 1;//ceil(CACHELINE/(double)iword);
// int   knsupc;    /* Size of supernode k.                               */
// int_t nlb;       /* Number of L blocks.                                */
// int_t  *ilsum = Llu->ilsum; /* Starting position of each supernode in lsum.   */

// int_t bid;
// int_t tmp;
// int_t tid = threadIdx_x + threadIdx_y * blockDim_x;
// int_t ready = 0;
// // int_t lock = 0;
// const int block_size = blockDim_x*blockDim_y; /* number of threads per block*/
// double zero = 0.0;


// double rC[THR_N][THR_M];

// gpuError_t error;

// bid= blockIdx_x;
// int_t idx = threadIdx_x;  // thread's m dimension
// int_t idy = threadIdx_y;  // thread's n dimension
// int_t ni,mi;


// int_t wrp;
// int_t lne = threadIdx_x & 0x1f ;
// // int_t ready = 0;
// // int_t lock = 0;
// const int warp_size = 32; /* number of threads per warp*/
// wrp= threadIdx_x + blockIdx_x * blockDim_x;
// wrp/=warp_size;



// // printf("  Entering kernel:   %i %i %i %i %i %i %i %i\n", threadIdx_x, blockIdx_x, grid->npcol, nsupers,myrow,krow,bid,tid);


// // rtemp_loc = (double*)malloc(maxsup*nrhs*Nbk*sizeof(double));

// if(wrp>=CEILING(nsupers, grid->npcol)){
// return;
// }else if(!Llu->Lrowind_bc_ptr[wrp]){
// return;
// }



// lk=wrp;
// iam = grid->iam;
// mycol = MYCOL( iam, grid );
// myrow = MYROW( iam, grid );
// k = mycol+lk*grid->npcol;
// knsupc = SuperSize( k );
// lsub = Llu->Lrowind_bc_ptr[lk];
// iam = grid->iam;
// krow = PROW( k, grid );
// lusup = Llu->Lnzval_bc_ptr[lk];
// lloc = Llu->Lindval_loc_bc_ptr[lk];
// nsupr = lsub[1];

// if(myrow==krow){
// nlb = lsub[0] - 1;
// idx_n = 1;
// idx_i = nlb+2;
// idx_v = 2*nlb+3;
// luptr_tmp = lloc[idx_v];
// m = nsupr-knsupc;
// }else{
// nlb = lsub[0];
// idx_n = 0;
// idx_i = nlb;
// idx_v = 2*nlb;
// luptr_tmp = lloc[idx_v];
// m = nsupr;
// }

// // printf("  Before kernel:   %i %i %i %i %i %i %i %i\n", threadIdx_x, blockIdx_x, grid->npcol, nsupers,myrow,krow,bid,tid);

// if(myrow==krow){   /* diagonal block performs trsm and forward the message*/

// if(lne==0){  /*only the first thread in a warp handles the lock */

// // printf("bk: %5d r: %5d %5d %5d\n",mycol+bid*grid->npcol,fmod[2*aln_i],myrow,krow);
// // for (i=0 ; i<maxsup ; i++){
// // rtemp_loc[i]=0.0;
// // }

// lib = LBi( k, grid ); /* Local block number, row-wise. */
// do{
// tmp=fmod[lib*aln_i];
// __threadfence();
// }while(tmp>0);

// }
// __syncwarp();


// lib = LBi( k, grid ); /* Local block number, row-wise. */
// il = LSUM_BLK( lib );
// ii = X_BLK( lib );

// for (i = lne; i < knsupc; i+=warp_size)
// x[i + ii ] += lsum[i + il ];
// // __syncwarp();


// if(Llu->inv == 1){
// Linv = Llu->Linv_bc_ptr[lk];
// for (i = lne; i < knsupc; i+=warp_size){
// temp1=zero;
// for (l=0 ; l<knsupc ; l++){
// temp1+=  Linv[l*knsupc+i]*x[ii+l];
// }
// lsum[il+i]=temp1; //reuse lsum as temporary output as it's no longer accessed
// }
// // __syncwarp();

// for (i = lne; i < knsupc; i+=warp_size){
// x[i + ii] = lsum[il+i];
// // printf("lk %5d %lf\n",lk,x[i + ii + j*knsupc]);
// }
// // __syncwarp();

// }
// // __syncwarp();
// }else{   /* off-diagonal block forward the message*/
// /* waiting for the x subvector and forward*/
// }


// if(nlb>0){
// if(nrhs==1){
// lib = LBi( k, grid ); /* Local block number, row-wise. */
// ii = X_BLK( lib );

// luptr_tmp1 = lloc[idx_v];
// lb = 0;
// nbrow=0;
// lptr1_tmp = lloc[lb+idx_i];
// lptr= lptr1_tmp+2;
// nbrow1 = lsub[lptr1_tmp+1];
// ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
// rel = xsup[ik]; /* Global row index of block ik. */
// lk = LBi( ik, grid ); /* Local block number, row-wise. */
// iknsupc = SuperSize( ik );
// il = LSUM_BLK( lk );
// for (i = lne; i < m; i+=warp_size){
// while(nbrow+lsub[lptr1_tmp+1]<=i){
// lb++;
// nbrow +=lsub[lptr1_tmp+1];
// lptr1_tmp = lloc[lb+idx_i];
// lptr= lptr1_tmp+2;
// ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
// rel = xsup[ik]; /* Global row index of block ik. */
// lk = LBi( ik, grid ); /* Local block number, row-wise. */
// iknsupc = SuperSize( ik );
// il = LSUM_BLK( lk );
// }

// irow = lsub[lptr+i-nbrow] - rel; /* Relative row. */
// RHS_ITERATE(j){
// temp1=zero;
// for (l=0 ; l<knsupc ; l++){
// temp1+= lusup[luptr_tmp1+l*nsupr+i]*x[ii+j*knsupc+l];
// }

// temp=atomicAdd(&lsum[il+irow + j*iknsupc],-temp1);
// }
// if(i==nbrow+lsub[lptr1_tmp+1]-1){
// fmod_tmp=atomicSub(&fmod[lk*aln_i],1);
// // __threadfence();
// }
// }
// }
// // __syncwarp();

// // if(tid==0){
// // for (lb = tid; lb < nlb; lb+=warp_size){
// // lptr1_tmp = lloc[lb+idx_i];
// // ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
// // lk = LBi( ik, grid ); /* Local block number, row-wise. */
// // fmod_tmp=atomicSub(&fmod[lk*aln_i],1);
// // // printf("k: %5d r: %5d\n",mycol+bid*grid->npcol,fmod[2*aln_i]);
// // }
// // }
// // __syncwarp();
// // } /*if tid<Nchunk*/
// } /* if nlb>0*/


// } /* dlsum_fmod_inv_gpu_1rhs */

__inline__ __device__
int warpReduceSum(int val) {
    for (int offset = warpSize/2; offset > 0; offset /= 2)
        //val += __shfl_down_sync(0xffffffff,val, offset,warpSize);
        val += __shfl_down_sync(0xffffffff, val, offset, warpSize);
    //__shfl_down_sync(unsigned mask, T var, unsigned int delta, int width=warpSize);
    return val;
}

__inline__ __device__
int warpAllReduceSum(int val) {
    for (int mask = warpSize/2; mask > 0; mask /= 2)
        val += __shfl_xor_sync(0xffffffff,val, mask,warpSize);
    return val;
}

__inline__ __device__
int blockReduceSum(int val, int bid, int tid, int mype) {

    static __shared__ int shared[32]; // Shared mem for 32 partial sums
    double sz=32.0;
    int lane = tid % warpSize;
    int wid = tid>>(int)log2(sz);
    val = warpReduceSum(val);     // Each warp performs partial reduction

    if (lane==0) shared[wid]=val; // Write reduced value to shared memory
    __syncthreads();              // Wait for all partial reductions

    //read from shared memory only if that warp existed
    val = (tid < (blockDim.x * blockDim.y) / warpSize) ? shared[lane] : 0;

    if (wid==0) val = warpReduceSum(val); //Final reduce within first warp

    return val;
}


__inline__ __device__ int warpReduceMin(int val)
{
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        int tmpVal = __shfl_down_sync(0xffffffff,val, offset, warpSize);
        if (tmpVal < val)  val = tmpVal;
    }
    return val;
}

__inline__ __device__  int blockReduceMin(int val,int bid, int tid, int mype)
{

    static __shared__ int shared[32]; // Shared mem for 32 partial mins
    double sz=32.0;
    int lane = tid % warpSize;
    int wid = tid>>(int)log2(sz);

    warpReduceMin(val);     // Each warp performs partial reduction

    if (lane == 0) shared[wid] = val; // Write reduced value to shared memory

    __syncthreads();              // Wait for all partial reductions

    //read from shared memory only if that warp existed
    val = (tid < (blockDim.x * blockDim.y) / warpSize) ? shared[lane] : INT_MAX;

    if (wid == 0)  warpReduceMin(val); //Final reduce within first warp
    return val;
}

/************************************************************************/
/*! \brief
 *
 * <pre>
 * Purpose
 * =======
 *   Perform local block modifications: lsum[i] -= L_i,k * X[k].
 * </pre>
 */
__global__ void dlsum_fmod_inv_gpu_mrhs_nvshmem
/************************************************************************/
        (
                int_t nbcol_loc,
                double *lsum,    /* Sum of local modifications.                        */
                double *x,       /* X array (local)                                    */
                int   nrhs,      /* Number of right-hand sides.                        */
                int   maxsup,      /* Max supernode size.                        */
                int_t   nsupers,      /* Number of total supernodes.                        */
                int_t *fmod,     /* Modification count for L-solve.                    */
                C_Tree  *LBtree_ptr,
                C_Tree  *LRtree_ptr,
                int_t *ilsum,
                int_t *Lrowind_bc_dat,
                long int *Lrowind_bc_offset,
                double *Lnzval_bc_dat,
                long int *Lnzval_bc_offset,
                double *Linv_bc_dat,
                long int *Linv_bc_offset,
                int_t *Lindval_loc_bc_dat,
                long int *Lindval_loc_bc_offset,
                int_t *xsup,
                gridinfo_t *grid,
                int_t maxrecvsz,
                int mype,
                volatile int* flag_bc_q,
                volatile int* flag_rd_q,
                double* ready_x,
                double* ready_lsum,
                int* my_flag_bc,
                int* my_flag_rd,
                int* d_launch_flag,
                int* d_nfrecv,
                volatile int* d_status,
                volatile int* d_statusmod,
                int_t nblock_ex
        )
{
    double alpha = 1.0, beta = 0.0,malpha=-1.0;
    double *lusup, *lusup1;
    double *dest;
    double *Linv;/* Inverse of diagonal block */
    int    iam, iknsupc, myrow, mycol, krow, nbrow, nbrow1, nbrow_ref, nsupr, nsupr1, p, pi, idx_r,m;
    int_t  k,i, l,ii,jj, ik, il, ikcol, irow, j, lb, lk, rel, lib,lready;
    int_t  *lsub, *lsub1, nlb1, lptr1, luptr1,*lloc;
    int_t  luptr_tmp,luptr_tmp1,lptr1_tmp, idx_i, idx_v,idx_n,  idx_l, fmod_tmp, lbstart,lbend,nn,Nchunk,nlb_loc,remainder;
    int thread_id1;
    flops_t ops_loc=0.0;
    MPI_Status status;
    int test_flag;
    yes_no_t done;
    int_t* idx_lsum,idx_lsum1;
    const int Nbk=1;
    __shared__ double rtemp_loc[128];
    double temp,temp1;
    int_t ldalsum;
    int_t nleaf_send_tmp;
    int_t lptr;      /* Starting position in lsub[*].                      */
    int_t luptr;     /* Starting position in lusup[*].                     */
    int_t iword = sizeof(int_t);
    int_t dword = sizeof (double);
    int_t aln_d,aln_i;
    aln_d = 1;//ceil(CACHELINE/(double)dword);
    aln_i = 1;//ceil(CACHELINE/(double)iword);
    int   knsupc;    /* Size of supernode k.                               */
    int_t nlb;       /* Number of L blocks.                                */

    int_t bid=blockIdx_x;
    int_t tmp;
    int_t tid = threadIdx_x + threadIdx_y * blockDim_x;
    int_t ready = 0;
    // int_t lock = 0;
    const int block_size = blockDim_x*blockDim_y; /* number of threads per warp*/
    double zero = 0.0;
    double rC[THR_N][THR_M];
    gpuError_t error;
    int_t idx = threadIdx_x;  // thread's m dimension
    int_t idy = threadIdx_y;  // thread's n dimension
    int_t ni,mi;
    int cnt;
    yes_no_t test;
    // rtemp_loc = (double*)malloc(maxsup*nrhs*Nbk*sizeof(double));
    //int delta=(nblock_ex < nbcol_loc ? nblock_ex : nbcol_loc);
    //int_t bid1;
    //bool flag=false;
    //if (bid<(delta*2)){
    //    bid1=bid/2;
    //    flag= bid%2==0 ? 1:0;
    //}else{
    //    bid1=bid-delta;
    //    flag= delta==nblock_ex ? 1:0;
    //}
    //if(tid==0) printf("(%d) iam bid=%d,bid1=%d,flag=%d\n",mype,bid,bid1,flag);
    //if(tid==0) printf("(%d) iam bid=%d,enter solve 0\n",mype,bid);

    //if(bid<nbcol_loc){
    if (Lrowind_bc_offset[bid] == -1) {
       return;
    }
    //if(tid==0) printf("(%d) iam bid=%d,enter solve--1\n",mype,bid);
    int get_offset, get_msgsize, get_rank, gc, gr, tmp_id, recv_offset = 0;

    lk = bid;
    iam = grid->iam;
    mycol = MYCOL(iam, grid);
    myrow = MYROW(iam, grid);
    gc = mycol + lk * grid->npcol;
    if (gc >= nsupers) return;
    k = gc; //mycol + lk * grid->npcol;

    knsupc = SuperSize(k);
    lsub = &Lrowind_bc_dat[Lrowind_bc_offset[lk]];
    iam = grid->iam;
    krow = PROW(k, grid);
    lusup = &Lnzval_bc_dat[Lnzval_bc_offset[lk]];
    lloc = &Lindval_loc_bc_dat[Lindval_loc_bc_offset[lk]];
    nsupr = lsub[1];

    if (myrow == krow) {
        nlb = lsub[0] - 1;
        idx_n = 1;
        idx_i = nlb + 2;
        idx_v = 2 * nlb + 3;
        luptr_tmp = lloc[idx_v];
        m = nsupr - knsupc;
    } else {
        nlb = lsub[0];
        idx_n = 0;
        idx_i = nlb;
        idx_v = 2 * nlb;
        luptr_tmp = lloc[idx_v];
        m = nsupr;
    }


    // printf("  Before kernel:   %i %i %i %i %i %i %i %i\n", threadIdx_x, blockIdx_x, grid->npcol, nsupers,myrow,krow,bid,tid);
    if (myrow == krow) {   /* diagonal block performs trsm and forward the message*/

        if (tid == 0) {  /*only the first thread in a block handles the lock */
            //printf("(%d) iam bid=%d,enter solve--2, wait lock,gc=%d\n",mype,bid,gc);
            //printf("bk: %5d r: %5d %5d %5d\n",mycol+bid*grid->npcol,fmod[2*aln_i],myrow,krow);
            // for (i=0 ; i<maxsup ; i++){
            // rtemp_loc[i]=0.0;
            // }

            lib = LBi(k, grid); /* Local block number, row-wise. */
            do {
                tmp = fmod[lib * aln_i];
                __threadfence();
            } while (tmp > 0);
        }
        __syncthreads();
        //if(tid==0) printf("(%d) iam bid=%d,enter solve--2, unlock,gc=%d\n",mype,bid,gc);

        lib = LBi(k, grid); /* Local block number, row-wise. */
        il = LSUM_BLK(lib);
        ii = X_BLK(lib);

        RHS_ITERATE(j)for (i = tid; i < knsupc; i += block_size) {
                //atomicAdd(&ready_x[0],lsum[i + il + j * knsupc]);
                x[i + ii + j * knsupc] += lsum[i + il + j * knsupc];
            }
        __syncthreads();
        //if(tid==0) printf("(%d,%d,%d),CHECKING k=%d,gc=%d,checksum=%lf\n",mype,bid,tid,k,gc,ready_x[0]);

        //  if(Llu->inv == 1){

        Linv = &Linv_bc_dat[Linv_bc_offset[lk]];

        if (nrhs == 1) {

            for (i = tid; i < knsupc; i += block_size) {
                temp1 = zero;
                for (l = 0; l < knsupc; l++) {
                    temp1 += Linv[l * knsupc + i] * x[ii + l];
                }
                lsum[il + i] = temp1; //reuse lsum as temporary output as it's no longer accessed
            }
            __syncthreads();

            for (i = tid; i < knsupc; i += block_size) {
                x[i + ii] = lsum[il + i];
                // printf("lk %5d %lf\n",lk,x[i + ii + j*knsupc]);
            }
            __syncthreads();

           // RHS_ITERATE(j){

           // for (i = tid; i < knsupc; i+=block_size)
           // rtemp_loc[i]=zero;
           // __syncthreads();


           // gemv_device_dlsum_fmod(
           // knsupc, knsupc, alpha,
           // Linv, knsupc,
           // &x[ii+j*knsupc], 1, beta,
           // rtemp_loc, 1);

           // __syncthreads();
           // // printf("tid %5d knsupc %5d block_size %5d\n",tid,knsupc,block_size);
           // for (i = tid; i < knsupc; i+=block_size){
           // x[i + ii + j*knsupc] = rtemp_loc[i];
           // // printf("lk %5d %lf\n",lk,x[i + ii + j*knsupc]);
           // }
           // }
           // __syncthreads();

        } else {
           __syncthreads();
           for (int_t blx = 0; blx * BLK_M < knsupc; blx++) {
               for (int_t bly = 0; bly * BLK_N < nrhs; bly++) {
                   gemm_device_dlsum_fmod(knsupc, nrhs, knsupc, blx, bly,
                                          Linv, knsupc, &x[ii], knsupc, rC,
                                          alpha, beta);
#pragma unroll
                   for (ni = 0; ni < THR_N; ni++) {
                       int_t coord_dCn = bly * BLK_N + ni * DIM_Y + idy;
#pragma unroll
                       for (mi = 0; mi < THR_M; mi++) {
                           int_t coord_dCm = blx * BLK_M + mi * DIM_X + idx;
                           if (coord_dCm < knsupc && coord_dCn < nrhs) {
                               double &regC = rC[ni][mi];
                               lsum[coord_dCm + il + coord_dCn *
                                                     knsupc] = regC;  //reuse lsum as temporary output as it's no longer accessed
                           }//if (coord_dCm < knsupc && coord_dCn < nrhs)
                       }
                   }
               }
           }
           __syncthreads();

           RHS_ITERATE(j)for (i = tid; i < knsupc; i += block_size)
                   x[i + ii + j * knsupc] = lsum[i + il + j * knsupc];
           __syncthreads();
        }//if(nrhs==1)

        RHS_ITERATE(j)for (i = tid; i < knsupc; i += block_size)
                ready_x[i + maxrecvsz * lk + j * knsupc] = x[i + ii + j * knsupc];

        __syncthreads();
    } else {   /* off-diagonal block forward the message*/
            /* waiting for the x subvector and forward*/
            //YL: only the first thread in a block spin-waits for the coming x subvector message using NVSHMEM, put the message into ready_x[maxrecvsz*lk]
        volatile int msg_recv = 0;
        if (tid == 0) {
            //printf("in solve WAIT1 (%d,%d) wait for col %d,flag=%d\n", mype, bid, gc,flag_bc_q[gc]);
            //nvshmem_int_wait_until((int *) flag_bc_q + gc, NVSHMEM_CMP_EQ, 1);
            do {
                msg_recv = flag_bc_q[lk];
                //msg_recv=d_status[gc];
                //msg_recv=flag_bc_q[gc];
                __threadfence();
            } while (msg_recv != 1);
            //printf("(%d,%d,%d,%d) in compute kernel, I have msg=%d,sz=%d.ofset=%d\n",mype,bid,tid,gc,msg_recv,LBtree_ptr[lk].msgSize_*nrhs+XK_H,maxrecvsz*lk);
            //double sum=0;
            //for (int myi=0;myi<LBtree_ptr[lk].msgSize_*nrhs+XK_H;myi++){
            //    sum+=ready_x[maxrecvsz*lk+myi];
            //}
            //printf("(%d,%d,%d), gc=%d,lk=%d, sum=%lf\n",mype,bid,tid,gc,lk,sum);
        }
        __syncthreads();
        //for(int i=0;i<LBtree_ptr[lk].msgSize_*nrhs+XK_H;i++){
        //    ready_x[maxrecvsz*lk+i]=ready_x[maxrecvsz*gc+i];
        //}
        //__syncthreads();
    }
    __syncthreads();

    //YL: only the first thread in a block forwards the x subvector using NVSHMEM
    cnt = LBtree_ptr[lk].destCnt_;
    if (cnt > 0) {
        //cnt=LBtree_ptr[lk].msgSize_;
        my_flag_bc[gc * RDMA_FLAG_SIZE] = lk;
        my_flag_bc[gc * RDMA_FLAG_SIZE + 1] = gc;
        my_flag_bc[gc * RDMA_FLAG_SIZE + 2] = maxrecvsz * lk;
        //my_flag_bc[gc*RDMA_FLAG_SIZE+2]=maxrecvsz*lk;
        my_flag_bc[gc * RDMA_FLAG_SIZE + 3] = LBtree_ptr[lk].msgSize_ * nrhs + XK_H;
        C_BcTree_forwardMessageSimple_Device(&LBtree_ptr[lk], (int *) flag_bc_q, &my_flag_bc[gc * RDMA_FLAG_SIZE],
                                             mype, tid, &ready_x[0]);
        //printf("(%d,%d,%d), lk=%d, gc=%d\n",mype,bid,tid,lk,gc);
        //C_BcTree_forwardMessageSimple_Device(&LBtree_ptr[lk],&ready_x[maxrecvsz*lk],cnt*nrhs+XK_H);
    }
    int keep_lk = lk;
    __syncthreads();

    if (nlb > 0) {

        lib = LBi(k, grid); /* Local block number, row-wise. */
        ii = X_BLK(lib);

        if (nrhs == 1) {
            luptr_tmp1 = lloc[idx_v];
            lb = 0;
            nbrow = 0;
            lptr1_tmp = lloc[lb + idx_i];
            lptr = lptr1_tmp + 2;
            nbrow1 = lsub[lptr1_tmp + 1];
            ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
            rel = xsup[ik]; /* Global row index of block ik. */
            lk = LBi(ik, grid); /* Local block number, row-wise. */
            iknsupc = SuperSize(ik);
            il = LSUM_BLK(lk);

            for (i = tid; i < m; i += block_size) {
                while (nbrow + lsub[lptr1_tmp + 1] <= i) {
                    lb++;
                    nbrow += lsub[lptr1_tmp + 1];
                    lptr1_tmp = lloc[lb + idx_i];
                    lptr = lptr1_tmp + 2;
                    ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
                    rel = xsup[ik]; /* Global row index of block ik. */
                    lk = LBi(ik, grid); /* Local block number, row-wise. */
                    iknsupc = SuperSize(ik);
                    il = LSUM_BLK(lk);
                }

                irow = lsub[lptr + i - nbrow] - rel; /* Relative row. */
                RHS_ITERATE(j) {
                    temp1 = zero;
                    for (l = 0; l < knsupc; l++) {
                        temp1 += lusup[luptr_tmp1 + l * nsupr + i] * ready_x[l + maxrecvsz * keep_lk + j * knsupc];
                    }

                    temp = atomicAdd(&lsum[il + irow + j * iknsupc], -temp1);
                    //printf("(%d,%d,%d),lsum[%d]=%f\n",mype,bid,tid,il+irow + j*iknsupc,lsum[il+irow + j*iknsupc]);
                }

                //  irow = lsub[lptr+i-nbrow] - rel; /* Relative row. */
                //  if(i==nbrow+lsub[lptr1_tmp+1]-1){
                // 	 fmod_tmp=atomicSub(&fmod[lk*aln_i],1);
                // 	 // __threadfence();
                //  }


            }
            __syncthreads();

            luptr_tmp1 = lloc[idx_v];
            lb = 0;
            nbrow = 0;
            lptr1_tmp = lloc[lb + idx_i];
            lptr = lptr1_tmp + 2;
            nbrow1 = lsub[lptr1_tmp + 1];
            ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
            rel = xsup[ik]; /* Global row index of block ik. */
            lk = LBi(ik, grid); /* Local block number, row-wise. */
            iknsupc = SuperSize(ik);
            il = LSUM_BLK(lk);
            gr=myrow + lk * grid->nprow;
            //knsupc = SuperSize(gr);

            for (i = tid; i < m; i += block_size) {
                while (nbrow + lsub[lptr1_tmp + 1] <= i) {
                    lb++;
                    nbrow += lsub[lptr1_tmp + 1];
                    lptr1_tmp = lloc[lb + idx_i];
                    lptr = lptr1_tmp + 2;
                    ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
                    rel = xsup[ik]; /* Global row index of block ik. */
                    lk = LBi(ik, grid); /* Local block number, row-wise. */
                    iknsupc = SuperSize(ik);
                    il = LSUM_BLK(lk);
                }
                //if (ik==15) printf("(%d) iam bid=%d,enter solve--3,fmod=%d\n",mype,bid,fmod_tmp);

                irow = lsub[lptr + i - nbrow] - rel; /* Relative row. */
                if (i == nbrow + lsub[lptr1_tmp + 1] - 1) {
                    // atomic return old val, omp return new val
                    fmod_tmp = atomicSub(&fmod[lk * aln_i], 1);
                    //printf("(%d) iam bid=%d,tid=%d,enter solve--6,i=%d,r=%d,lk=%d,ik=%d,fmod_tmp=%d,fmod=%d\n",mype,bid,tid,i,nbrow + lsub[lptr1_tmp + 1] - 1, lk,ik,fmod_tmp,fmod[lk * aln_i]);
                    // __threadfence();
                    if(fmod_tmp==1) {// forward RD
                        //senddone[lk]=1;
                        if(LRtree_ptr[lk].myRoot_ != LRtree_ptr[lk].myRank_){
                            //cnt=LRtree_ptr[lib].msgSize_;
                            my_flag_rd[ik*RDMA_FLAG_SIZE]=lk;
                            my_flag_rd[ik*RDMA_FLAG_SIZE+1]=LRtree_ptr[lk].msgSize_;
                            RHS_ITERATE(j) {
                                for (int aab = 0; aab < iknsupc; aab++) {
                                    ready_lsum[lk * maxrecvsz * 2 + aab +j * iknsupc] = lsum[il + aab +j * iknsupc];
                                    //printf("data3-(%d,%d,%d),lib=%d,k=%d,i=%d,ready_lsum[%d]=%f\n", mype, bid, tid, lib, k, i,
                                    //       k * maxrecvsz * 2 + i +j * knsupc,
                                    //       ready_lsum[k * maxrecvsz * 2 + i +j * knsupc]);

                                }
                            }
                            //printf("(%d,%d,%d) in solve,lib=%d,gr=%d,ik=%d,myflagrd=%d,%d\n",mype,bid,tid,lk,gr,ik,my_flag_rd[ik*RDMA_FLAG_SIZE],my_flag_rd[ik*RDMA_FLAG_SIZE+1]);
                            C_RdTree_forwardMessageSimple_Device(&LRtree_ptr[lk], (int*)flag_rd_q, &my_flag_rd[RDMA_FLAG_SIZE*ik], mype, bid, tid, &ready_lsum[0],maxrecvsz);
                        }
                    }
                }
            }
            //__syncthreads();

        } else {
            for (lb = 0; lb < nlb; lb++) {
                luptr_tmp1 = lloc[lb + idx_v];

                // nbrow=0;
                // lptr1_tmp = lloc[lb+idx_i];
                // nbrow += lsub[lptr1_tmp+1];


                lib = LBi(k, grid); /* Local block number, row-wise. */
                ii = X_BLK(lib);

                lptr1_tmp = lloc[lb + idx_i];
                lptr = lptr1_tmp + 2;
                nbrow1 = lsub[lptr1_tmp + 1];
                ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
                rel = xsup[ik]; /* Global row index of block ik. */

                lk = LBi(ik, grid); /* Local block number, row-wise. */

                iknsupc = SuperSize(ik);
                il = LSUM_BLK(lk);


                // if(nrhs==1){

                // for (i = tid; i < nbrow1; i+=block_size)
                // rtemp_loc[i]=zero;
                // __syncthreads();


                // gemv_device_dlsum_fmod(
                // nbrow1, knsupc, alpha,
                // &lusup[luptr_tmp1], nsupr,
                // &x[ii], 1, beta,
                // rtemp_loc, 1);

                // __syncthreads();
                // for (i = tid; i < nbrow1; i+=block_size){
                // irow = lsub[lptr+i] - rel; /* Relative row. */
                // temp=atomicAdd(&lsum[il+irow],-rtemp_loc[i]);
                // }
                // }else{

                for (int_t blx = 0; blx * BLK_M < nbrow1; blx++) {
                    for (int_t bly = 0; bly * BLK_N < nrhs; bly++) {
                        gemm_device_dlsum_fmod(nbrow1, nrhs, knsupc, blx, bly,
                                               &lusup[luptr_tmp1], nsupr, &ready_x[maxrecvsz * keep_lk], knsupc, rC,
                                               alpha, beta);
#pragma unroll
                            for (ni = 0; ni < THR_N; ni++) {
                                int_t coord_dCn = bly * BLK_N + ni * DIM_Y + idy;
#pragma unroll
                                for (mi = 0; mi < THR_M; mi++) {
                                    int_t coord_dCm = blx * BLK_M + mi * DIM_X + idx;
                                    if (coord_dCm < nbrow1 && coord_dCn < nrhs) {
                                        irow = lsub[lptr + coord_dCm] - rel; /* Relative row. */
                                        double &regC = rC[ni][mi];
                                        temp = atomicAdd(&lsum[il + irow + coord_dCn * iknsupc], -regC);
                                    }
                                }
                            }
                        }
                    }
                    // }//if(nrhs==1)

                    if (tid == 0)fmod_tmp = atomicSub(&fmod[lk * aln_i], 1);


                }

            }//if(nrhs==1)


            // if(tid==0){
            // for (lb = tid; lb < nlb; lb+=block_size){
            // lptr1_tmp = lloc[lb+idx_i];
            // ik = lsub[lptr1_tmp]; /* Global block number, row-wise. */
            // lk = LBi( ik, grid ); /* Local block number, row-wise. */
            // fmod_tmp=atomicSub(&fmod[lk*aln_i],1);
            // // printf("k: %5d r: %5d\n",mycol+bid*grid->npcol,fmod[2*aln_i]);
            // }
            // }
            //__syncthreads();
            // } /*if tid<Nchunk*/
        } /* if nlb>0*/
} /* dlsum_fmod_inv_gpu_mrhs */

__global__ void test(int nbcol_loc, int nblock_ex,int mype){
    int_t tid = threadIdx_x + threadIdx_y * blockDim_x;
    int_t bid= blockIdx_x;
    if(bid<nbcol_loc){
        if(tid==0) printf("(%d) iam in test if bid %d\n",mype,bid);

    }else if (bid<nbcol_loc + nblock_ex){
        if(tid==0) printf("(%d) iam bid %d\n",mype,bid);
    }
}

void dlsum_fmod_inv_gpu_wrap
        (
                int_t nbcol_loc,    /*number of local supernode columns*/
                int_t nbrow_loc,    /*number of local supernode rows*/
                int_t nthread_x,     /*kernel launch parameter*/
                int_t nthread_y,     /*kernel launch parameter*/
                double *lsum,    /* Sum of local modifications.                        */
                double *x,       /* X array (local)                                    */
                int   nrhs,      /* Number of right-hand sides.                        */
                int   maxsup,      /* Max supernode size.                        */
                int_t   nsupers,      /* Number of total supernodes.                        */
                int_t *fmod,     /* Modification count for L-solve.                    */
                C_Tree  *LBtree_ptr,
                C_Tree  *LRtree_ptr,
                int_t *ilsum,
                int_t *Lrowind_bc_dat,
                long int *Lrowind_bc_offset,
                double *Lnzval_bc_dat,
                long int *Lnzval_bc_offset,
                double *Linv_bc_dat,
                long int *Linv_bc_offset,
                int_t *Lindval_loc_bc_dat,
                long int *Lindval_loc_bc_offset,
                int_t *xsup,
                gridinfo_t *grid,
                int_t maxrecvsz,
                int* flag_bc_q,
                int* flag_rd_q,
                double* ready_x,
                double* ready_lsum,
                int* my_flag_bc,
                int* my_flag_rd,
                int* d_launch_flag,
                int* d_nfrecv,
                int* h_nfrecv,
                int* d_status,
                int* d_colnum,
                int* d_mynum,
                int* d_mymaskstart,
                int* d_mymasklength,
                int* d_nfrecvmod,
                int* d_statusmod,
                int* d_colnummod,
                int* d_mynummod,
                int* d_mymaskstartmod,
                int* d_mymasklengthmod,
                int* d_recv_cnt,
                int* d_msgnum,
                int* senddone
                //int* d_rownum,
                //int* d_rowstart,
                //int* d_validrows
        ) {

    gpuStream_t sid = 0;
    int gid = 0;
    int mycol;
    int_t lk, k, knsupc;
    //int_t nblock_ex=CEILING( nbrow_loc, nthread_x*nthread_y);
    int_t nblock_ex = CEILING(nbrow_loc, ((nthread_x * nthread_y) / 32)); //32 (warp) * 8 =256
    //int_t nblock_ex = nbrow_loc; //CEILING(nbrow_loc, ((nthread_x * nthread_y) / 32)); //32 (warp) * 8 =256
    //int_t nblock_ex = CEILING(nbrow_loc, ((nthread_x * nthread_y) / 64)); //32 (warp) * 8 =256

    int mype, npes, ndevices;
    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();
    //printf("(%d) nbcol_loc %d\n", mype, nbcol_loc);
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    CUDA_CHECK(hipSetDevice(mype_node));


    //printf("(%d), Enter,mynode=%d\n",mype,mype_node);
    //fflush(stdout);


    hipStream_t stream[2];
    for (int i = 0; i < 2; ++i) {
        //hipStreamCreate(&stream[i]);
        hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking);
    }
    //int minGridSize;
    //int myblockSize;
    //hipOccupancyMaxPotentialBlockSize(&minGridSize,&myblockSize,(const void *) schedule ,0,0 );
    //h_nfrecv[1]=myblockSize;
    //gpuMemcpy(d_nfrecv, h_nfrecv, 3 * sizeof(int), gpuMemcpyHostToDevice);
    //printf("(%d) solve=%d,%d, wait=%d,%d\n",mype,nbcol_loc,nthread_x*nthread_y,h_nfrecv[2],h_nfrecv[1]);
    //fflush(stdout);

    dim3 dimGrid_nv(h_nfrecv[2]); //2
    dim3 dimBlock_nv(h_nfrecv[1]); //1024
    dim3 dimGrid(nbcol_loc);
    dim3 dimBlock(nthread_x, nthread_y);

    int launch_success = 0;

    void *args[] = {&nrhs, &LRtree_ptr, &maxrecvsz, &mype, &flag_bc_q, &flag_rd_q,
                    &ready_x, &ready_lsum, &my_flag_bc, &my_flag_rd, &d_nfrecv, &d_status, &d_launch_flag,
                    &d_colnum, &d_mynum, &d_mymaskstart, &d_mymasklength,
                    &d_nfrecvmod, &d_statusmod, &d_colnummod, &d_mynummod, &d_mymaskstartmod, &d_mymasklengthmod,
                    &d_recv_cnt, &d_msgnum,&lsum,&fmod,&grid,&xsup,&ilsum,&nbrow_loc,&nsupers};

    NVSHMEM_CHECK(
            nvshmemx_collective_launch((const void *) schedule, dimGrid_nv, dimBlock_nv, args, 0, stream[0]));
    do {
        hipMemcpyAsync(&launch_success, d_launch_flag, 1 * sizeof(int), hipMemcpyDeviceToHost, stream[1]);
    } while (launch_success == 0);


    if (launch_success == 1)
    dlsum_fmod_inv_gpu_mrhs_nvshmem<<< dimGrid, dimBlock, 0, stream[1] >>>(nbcol_loc,
                                                                           lsum, x,
                                                                           nrhs, maxsup, nsupers,
                                                                           fmod,
                                                                           LBtree_ptr, LRtree_ptr,
                                                                           ilsum,
                                                                           Lrowind_bc_dat,
                                                                           Lrowind_bc_offset,
                                                                           Lnzval_bc_dat,
                                                                           Lnzval_bc_offset,
                                                                           Linv_bc_dat,
                                                                           Linv_bc_offset,
                                                                           Lindval_loc_bc_dat,
                                                                           Lindval_loc_bc_offset,
                                                                           xsup,
                                                                           grid, maxrecvsz,
                                                                           mype, flag_bc_q,
                                                                           flag_rd_q,
                                                                           ready_x, ready_lsum,
                                                                           my_flag_bc, my_flag_rd,
                                                                           d_launch_flag,
                                                                           d_nfrecv, d_status,
                                                                           d_statusmod,nblock_ex);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

}


 /************************************************************************/
 /*! \brief
  *
  * <pre>
  * Purpose
  * =======
  *   Perform local block modifications: lsum[i] -= L_i,k * X[k].
  * </pre>
  */
  __global__ void dlsum_bmod_inv_gpu_mrhs
  /************************************************************************/
  (
   int_t nbcol_loc,
   double *lsum,    /* Sum of local modifications.                        */
   double *x,       /* X array (local)                                    */
   int   nrhs,      /* Number of right-hand sides.                        */
   int_t   nsupers,      /* Number of total supernodes.                        */
   int_t *bmod,     /* Modification count for U-solve.                    */
   C_Tree  *UBtree_ptr,
   C_Tree  *URtree_ptr,
   int_t *ilsum,
  int_t *Urbs,
  int_t   *Ufstnz_br_dat,
  long int *Ufstnz_br_offset,
  double *Unzval_br_dat,  
  long int *Unzval_br_offset,
  int_t  *Ucb_valdat,    
  long int *Ucb_valoffset,  
  Ucb_indptr_t *Ucb_inddat,
  long int *Ucb_indoffset,
  double *Uinv_bc_dat,     
  long int *Uinv_bc_offset,   
  int_t *xsup,
  gridinfo_t *grid
  )
  {
	  double alpha = 1.0, beta = 0.0,malpha=-1.0;
	  double xtemp;
	  double *lusup, *lusup1;
	  double *dest;
	  double *Uinv;/* Inverse of diagonal block */
	  int    iam, iknsupc, myrow, mycol, krow, nbrow, nbrow1, nbrow_ref, nsupr, nsupr1, p, pi, idx_r,m;
	  int_t  k,i, l,ii,jj, ik, il, ikcol, irow, j, lb, lk, rel, lib,lready, ub;
	  int_t  *lsub, *lsub1, nlb1, lptr1, luptr1,*lloc;
	  int_t  luptr_tmp,luptr_tmp1,lptr1_tmp, idx_i, idx_v,idx_n,  idx_l, fmod_tmp, lbstart,lbend,nn,Nchunk,nlb_loc,remainder;
	  int thread_id1;
	  int_t gik,ikfrow,iklrow;
	  int_t  uptr;
	  int_t fnz,fnzmin;
	  flops_t ops_loc=0.0;
	  MPI_Status status;
	  int test_flag;
	  yes_no_t done;
	  int_t* idx_lsum,idx_lsum1;
	  const int Nbk=1;
	//   __shared__ double rtemp_loc[128]; 
	  double temp,temp1;
	  double temp2[MAXSUPER];
	//   int_t temp3[128];
	//   int_t temp4[128];
	  int_t ldalsum;
	  int_t nleaf_send_tmp;
	  int_t lptr;      /* Starting position in lsub[*].                      */
	  int_t luptr;     /* Starting position in lusup[*].                     */
	  int_t iword = sizeof(int_t);
	  int_t dword = sizeof (double);
	  int_t aln_d,aln_i;
	  aln_d = 1;//ceil(CACHELINE/(double)dword);
	  aln_i = 1;//ceil(CACHELINE/(double)iword);
	  int   knsupc;    /* Size of supernode k.                               */
	  int_t nub;       /* Number of L blocks.                                */
 
	  int_t bid;
	  int_t tmp;
	  int_t tid = threadIdx_x + threadIdx_y * blockDim_x; 
	  int_t ready = 0;
	  // int_t lock = 0;
	  const int block_size = blockDim_x*blockDim_y; /* number of threads per warp*/
	  double zero = 0.0;
	  	
  
	  double rC[THR_N][THR_M];
	  
	  gpuError_t error;
	  
	  bid= nbcol_loc-blockIdx_x-1;  // This makes sure higher block IDs are checked first in spin wait
	  int_t idx = threadIdx_x;  // thread's m dimension
	  int_t idy = threadIdx_y;  // thread's n dimension
	  int_t ni,mi;
	  int cnt;
	  yes_no_t test;

	  int_t  *usub;
	  double *uval, *y;

	  
	  
	  // printf("  Entering kernel:   %i %i %i %i %i %i %i %i\n", threadIdx_x, blockIdx_x, grid->npcol, nsupers,myrow,krow,bid,tid);
	  
	  
	  // rtemp_loc = (double*)malloc(maxsup*nrhs*Nbk*sizeof(double));
	  
	  
	  // the first nbcol_loc handles all computations and broadcast communication
	  if(bid<nbcol_loc){
		  if(Uinv_bc_offset[bid]==-1){
		  return;
		  }
		  
		  lk=bid;
		  iam = grid->iam;
		  mycol = MYCOL( iam, grid );
		  myrow = MYROW( iam, grid );
		  k = mycol+lk*grid->npcol;
		  knsupc = SuperSize( k );
		  krow = PROW( k, grid );	
		  nub = Urbs[lk];      /* Number of U blocks in block column lk */
		  
		//   printf("  Before kernel:   %i %i %i %i %i %i %i %i\n", threadIdx_x, blockIdx_x, grid->npcol, nsupers,myrow,krow,bid,tid);
		  
		  if(myrow==krow){   /* diagonal block performs trsm and forward the message*/
  
			  if(tid==0){  /*only the first thread in a block handles the lock */
  
			  
			  // for (i=0 ; i<maxsup ; i++){
				  // rtemp_loc[i]=0.0;
			  // }	
			  
				  lib = LBi( k, grid ); /* Local block number, row-wise. */
				//   printf("bk: %5d r: %5d %5d %5d\n",mycol+bid*grid->npcol,bmod[lib*aln_i],myrow,krow);
				  do{
					  tmp=bmod[lib*aln_i];
					  __threadfence();			
				  }while(tmp>0);
				  
			  }
			  __syncthreads();
			//   if(tid==0)
			//   printf("spin: %d %d \n",threadIdx_x, blockIdx_x);

				  
				  lib = LBi( k, grid ); /* Local block number, row-wise. */
				  il = LSUM_BLK( lib );
				  ii = X_BLK( lib );
				  
				  RHS_ITERATE(j)
					  for (i = tid; i < knsupc; i+=block_size)
						  x[i + ii + j*knsupc] += lsum[i + il + j*knsupc ];
				  __syncthreads();
				  
				  
				 //  if(Llu->inv == 1){
				  
					  Uinv = &Uinv_bc_dat[Uinv_bc_offset[lk]];
						  
					  if(nrhs==1){
						  for (i = tid; i < knsupc; i+=block_size){					
							  temp1=zero;
							  for (l=0 ; l<knsupc ; l++){
								  temp1+=  Uinv[l*knsupc+i]*x[ii+l];
							  }								
							  lsum[il+i]=temp1; //reuse lsum as temporary output as it's no longer accessed
						  }
						  __syncthreads();					
							  
						  for (i = tid; i < knsupc; i+=block_size){
							  x[i + ii] = lsum[il+i];
							  // printf("lk %5d %lf\n",lk,x[i + ii + j*knsupc]);
							  }					
						  __syncthreads();		
					  }else{
						  __syncthreads(); 	
						  for (int_t blx = 0; blx*BLK_M < knsupc; blx++){
							  for (int_t bly = 0; bly*BLK_N < nrhs; bly++){
								  gemm_device_dlsum_fmod(knsupc, nrhs, knsupc, blx, bly, 
								  Uinv, knsupc, &x[ii], knsupc, rC,
								  alpha, beta);
									  #pragma unroll
								  for (ni = 0; ni < THR_N; ni++) {
									  int_t coord_dCn = bly*BLK_N + ni*DIM_Y + idy;
									  #pragma unroll
									  for (mi = 0; mi < THR_M; mi++) {
										  int_t coord_dCm = blx*BLK_M + mi*DIM_X + idx;
										  if (coord_dCm < knsupc && coord_dCn < nrhs) {
											  double &regC = rC[ni][mi];
											  lsum[coord_dCm + il + coord_dCn*knsupc ]=regC;  //reuse lsum as temporary output as it's no longer accessed
										  }//if (coord_dCm < knsupc && coord_dCn < nrhs)
									  }
								  }						
							  }
						  }
						  __syncthreads(); 	
  
						  RHS_ITERATE(j)
						  for (i = tid; i < knsupc; i+=block_size)
							  x[i + ii + j*knsupc] = lsum[i + il + j*knsupc ];
						  __syncthreads(); 		
					  }//if(nrhs==1)
				 //  }
				  
				//   RHS_ITERATE(j)
				//   for (i = tid; i < knsupc; i+=block_size)
				// 	  recvbuf_BC_gpu[i + maxrecvsz*lk + j*knsupc ] = x[i + ii + j*knsupc];
					  
			  __syncthreads();	
		  }else{   /* off-diagonal block forward the message*/
			  /* waiting for the x subvector and forward*/ 
			  if(tid==0){  //YL: only the first thread in a block spin-waits for the coming x subvector message using NVSHMEM, put the message into recvbuf_BC_gpu[maxrecvsz*lk]
			  
			  }
		  }
		   
		  
		//   if(tid==0){  //YL: only the first thread in a block forwards the x subvector using NVSHMEM
		//   cnt=LBtree_ptr[lk].destCnt_;
		//  //  printf("good1 %5d%5d\n",lk,cnt);
		//   if(cnt>0){
		// 	 cnt=LBtree_ptr[lk].msgSize_;
		// 	  C_BcTree_forwardMessageSimple_Device(&LBtree_ptr[lk],&recvbuf_BC_gpu[maxrecvsz*lk],cnt*nrhs+XK_H);
		//   }
		//   }	
		  
		  if(nub>0){
		  
				  lib = LBi( k, grid ); /* Local block number, row-wise. */
				  ii = X_BLK( lib );	


					int ngroup=SUPERLU_MIN(nub,block_size);
					int block_size_loc = floor((double)block_size/ngroup);
					int remainder = nub % ngroup;
					int gid=tid/block_size_loc;

					for (ub = gid; ub < nub; ub+=ngroup) {
						ik = Ucb_inddat[Ucb_indoffset[lk]+ub].lbnum; /* Local block number, row-wise. */
						usub = &Ufstnz_br_dat[Ufstnz_br_offset[ik]];
						uval = &Unzval_br_dat[Unzval_br_offset[ik]];
						i = Ucb_inddat[Ucb_indoffset[lk]+ub].indpos; /* Start of the block in usub[]. */
						i += UB_DESCRIPTOR;
						il = LSUM_BLK( ik );
						gik = ik * grid->nprow + myrow;/* Global block number, row-wise. */
						iknsupc = SuperSize( gik );
						ikfrow = FstBlockC( gik );
						iklrow = FstBlockC( gik+1 );
			
						// printf("ub %d bmod: %d \n",ub, bmod[ik*aln_i]);

						if(tid % block_size_loc==0){ // parallelizing this supernode across knsupc or irow doesn't seem to have any benefit
						fnzmin=100000000;
						for (jj = 0; jj < knsupc; ++jj)
							fnzmin = min(fnzmin,usub[i + jj]);
						RHS_ITERATE(j) {
							dest = &lsum[il + j*iknsupc];
							uptr = Ucb_valdat[Ucb_valoffset[lk]+ub]; /* Start of the block in uval[]. */
							
							for (jj = 0; jj < iknsupc; ++jj) 
								temp2[jj]=0;
							for (jj = 0; jj < knsupc; ++jj) {
								fnz = usub[i + jj];
								if ( fnz < iklrow ) { /* Nonzero segment. */
									/* AXPY */
									xtemp=x[ii+j*knsupc+jj];
									for (irow = fnz; irow < iklrow; ++irow){
										temp2[irow - ikfrow]+=uval[uptr++] * xtemp; // YL: this is most expensive operation on GPU
									}
								}
							} /* for jj ... */

							for (irow = fnzmin; irow < iklrow; ++irow){
								temp=atomicAdd(&dest[irow - ikfrow],-temp2[irow - ikfrow]);
							}								
						}
						fmod_tmp=atomicSub(&bmod[ik*aln_i],1);
						}
						
					} /* for ub ... */
				// }
				__syncthreads();

			
				//   __syncthreads();
			  // } /*if tid<Nchunk*/
		  } /* if nlb>0*/		
  
		  // printf("nimbgood \n");
  
//   }else if(bid<nbcol_loc+nblock_ex){  //the next nblock_ex blocks handle all reduction communication
	  
  }
  
		  
	  
  } /* dlsum_bmod_inv_gpu_mrhs */
  
  
 


 

 





 void dlsum_fmod_inv_gpu_wrap
 (
  int_t nbcol_loc,    /*number of local supernode columns*/
  int_t nbrow_loc,    /*number of local supernode rows*/
  int_t nthread_x,     /*kernel launch parameter*/
  int_t nthread_y,     /*kernel launch parameter*/
  double *lsum,    /* Sum of local modifications.                        */
  double *x,       /* X array (local)                                    */
  int   nrhs,      /* Number of right-hand sides.                        */
  int   maxsup,      /* Max supernode size.                        */
  int_t   nsupers,      /* Number of total supernodes.                        */
  int_t *fmod,     /* Modification count for L-solve.                    */
  C_Tree  *LBtree_ptr,
  C_Tree  *LRtree_ptr,
  int_t *ilsum,
  int_t *Lrowind_bc_dat,   
  long int *Lrowind_bc_offset,      
  double *Lnzval_bc_dat,     
  long int *Lnzval_bc_offset,     
  double *Linv_bc_dat,     
  long int *Linv_bc_offset,     
  int_t *Lindval_loc_bc_dat,     
  long int *Lindval_loc_bc_offset,     
  int_t *xsup,
  gridinfo_t *grid,
  double * recvbuf_BC_gpu,
  double * recvbuf_RD_gpu,
  int_t maxrecvsz
 ){
 
 gpuStream_t sid=0;
 int gid=0;
 int mycol;
 int_t lk,k,knsupc;
 int_t nblock_ex=CEILING( nbrow_loc, nthread_x*nthread_y);;
 
	 
	 // printf("pinv %d\n",Llu->inv);
	 // fflush(stdout);

 
	 // if(nrhs>1){
		 dim3 dimBlock(nthread_x, nthread_y);
		 dlsum_fmod_inv_gpu_mrhs<<< nbcol_loc+nblock_ex, dimBlock >>>(nbcol_loc,nblock_ex,lsum,x,nrhs,maxsup,nsupers,fmod,LBtree_ptr,LRtree_ptr,ilsum,Lrowind_bc_dat,Lrowind_bc_offset,Lnzval_bc_dat,Lnzval_bc_offset,Linv_bc_dat,Linv_bc_offset,Lindval_loc_bc_dat,Lindval_loc_bc_offset, xsup,grid,recvbuf_BC_gpu,recvbuf_RD_gpu,maxrecvsz);
	 // }else{
		 // dim3 dimBlock(nthread_x*nthread_y, 1);
		 // dlsum_fmod_inv_gpu_1rhs<<< CEILING(nbcol_loc,NWARP), dimBlock >>>(lsum,x,rtemp,nrhs,maxsup,nsupers,fmod,xsup,grid,Llu);	
	 // }


 
	 gpuDeviceSynchronize();
 }




 void dlsum_bmod_inv_gpu_wrap
 (
  int_t nbcol_loc,    /*number of local supernode columns*/
  int_t nbrow_loc,    /*number of local supernode rows*/
  int_t nthread_x,     /*kernel launch parameter*/
  int_t nthread_y,     /*kernel launch parameter*/
  double *lsum,    /* Sum of local modifications.                        */
  double *x,       /* X array (local)                                    */
  int   nrhs,      /* Number of right-hand sides.                        */
  int   maxsup,      /* Max supernode size.                        */
  int_t   nsupers,      /* Number of total supernodes.                        */
  int_t *bmod,     /* Modification count for L-solve.                    */
  C_Tree  *UBtree_ptr,
  C_Tree  *URtree_ptr,
  int_t *ilsum,
  int_t *Urbs,
  int_t   *Ufstnz_br_dat,
  long int *Ufstnz_br_offset,
  double *Unzval_br_dat,  
  long int *Unzval_br_offset,
  int_t  *Ucb_valdat,    
  long int *Ucb_valoffset,  
  Ucb_indptr_t *Ucb_inddat,
  long int *Ucb_indoffset,
  double *Uinv_bc_dat,     
  long int *Uinv_bc_offset,   
  int_t *xsup,
  gridinfo_t *grid
 ){
 
 gpuStream_t sid=0;
 int gid=0;
 int mycol;
 int_t lk,k,knsupc;

	 
	 // printf("pinv %d\n",Llu->inv);
	 // fflush(stdout);
		int_t maxsuper = sp_ienv_dist(3);
		if(MAXSUPER<maxsuper){
			printf("increase MAXSUPER\n");
			exit(1);
		}
		dim3 dimBlock(nthread_x, nthread_y);
		dlsum_bmod_inv_gpu_mrhs<<< nbcol_loc, dimBlock >>>(nbcol_loc,lsum,x,nrhs,nsupers,bmod, UBtree_ptr,URtree_ptr,ilsum,Urbs,Ufstnz_br_dat,Ufstnz_br_offset,Unzval_br_dat,Unzval_br_offset,Ucb_valdat,Ucb_valoffset,Ucb_inddat,Ucb_indoffset,Uinv_bc_dat,Uinv_bc_offset,xsup,grid);



 
	 gpuDeviceSynchronize();
 }

 
 
#ifdef __cplusplus
}
#endif
