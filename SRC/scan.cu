#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>

// typedef float pfx_dtype ; 
typedef int pfx_dtype ; 
__global__ void prescan(pfx_dtype *outArr, pfx_dtype *inArr, int n)
{
    extern __shared__ pfx_dtype temp[];
    int thread_id = threadIdx.x;
    int offset = 1;
    temp[2*thread_id] = inArr[2*thread_id]; 
    temp[2*thread_id+1] = inArr[2*thread_id+1];
    for (int d = n>>1; d > 0; d >>= 1) 
    {
        __syncthreads();
        if (thread_id < d)
        {
            int ai = offset*(2*thread_id+1)-1;
            int bi = offset*(2*thread_id+2)-1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }
    
    if (thread_id == 0) { temp[n - 1] = 0; } 
    for (int d = 1; d < n; d *= 2) 
    {
        offset >>= 1;
        __syncthreads();
        if (thread_id < d)
        {
            int ai = offset*(2*thread_id+1)-1;
            int bi = offset*(2*thread_id+2)-1;
            pfx_dtype t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();
    outArr[2*thread_id] = temp[2*thread_id]+ inArr[2*thread_id]; // write results to device memory
    outArr[2*thread_id+1] = temp[2*thread_id+1]+ inArr[2*thread_id+1];
    __syncthreads();
    printf("xA[%d] = %d \n",2*thread_id , outArr[2*thread_id]);
    printf("xA[%d] = %d \n",2*thread_id+1 , outArr[2*thread_id+1]);
    __syncthreads();
} 

#define SELF_TEST 
#ifdef SELF_TEST

#include <iostream>
#include "hipcub/hipcub.hpp"
#define N 22
#define THREAD_BLOCK_SIZE 32


__global__
void cub_scan_test(void)
{
	int thread_id = threadIdx.x;
	typedef hipcub::BlockScan<int, THREAD_BLOCK_SIZE > BlockScan; /*1D int data type*/

	__shared__ typename BlockScan::TempStorage temp_storage; /*storage temp*/

	__shared__ int IndirectJ1[N];
	__shared__ int IndirectJ2[N];

	if (thread_id < N)
	{
		IndirectJ1[thread_id] = 2*thread_id +1;
	}

	__syncthreads();
	if (thread_id < THREAD_BLOCK_SIZE)
		BlockScan(temp_storage).InclusiveSum (IndirectJ1[thread_id], IndirectJ2[thread_id]);


	if (thread_id < THREAD_BLOCK_SIZE)
		printf("%d %d\n", thread_id, IndirectJ2[thread_id]);

}



// extern __shared__
// #define THREAD_BLOCK_SIZE 7

__global__ void initData(pfx_dtype* A, int n)
{
    int threadId = threadIdx.x;   
    if(threadId<n)
        A[threadId] = 2*threadId+1;
        printf("A[%d] = %d \n",threadId,A[threadId]);
}

int main()
{
    
    pfx_dtype *A, *xA;
    hipMalloc(&A, sizeof(pfx_dtype)*N);
    hipMalloc(&xA, sizeof(pfx_dtype)*N);
    

    initData<<< 1,THREAD_BLOCK_SIZE >>> (A,N);
    if(hipDeviceSynchronize() != hipSuccess)
        std::cout<<"Error- 0\n";
    // prescan<<<  1,THREAD_BLOCK_SIZE/2,2*THREAD_BLOCK_SIZE*sizeof(pfx_dtype) >>> (xA, A, N);
    prescan<<<  1,(N+1)/2,2*N*sizeof(pfx_dtype) >>> (xA, A, N);
    if(hipDeviceSynchronize() != hipSuccess)
        std::cout<<".....EXITING\n";   
    else
        std::cout<<"No errors reported\n";


    // typedef hipcub::BlockScan<int, THREAD_BLOCK_SIZE> BlockScan; /*1D int data type*/
	// __shared__ typename BlockScan::TempStorage temp_storage; /*storage temp*/

    cub_scan_test <<<  1,THREAD_BLOCK_SIZE >>> ();

    return 0;
}

#endif 